#include "hip/hip_runtime.h"
#include "opencv_chapter10.cuh"
//һ������任
//�����init=0����Hough_mat,init=1
Mat Hough_gpu(Mat& image_N)//Hough_mat����������
{  
	Mat image = image_N.clone();
	image.convertTo(image, CV_32F);

	int angle_min = -90;//����
	int angle_max = 90;

	int P_min = (int)-sqrt(pow(image.rows, 2.0) + pow(image.cols, 2.0));//����
	int P_max = (-1.0)*P_min;

	int p_len = P_max+1;
	int a_len = angle_max+1;

	Mat Hough_mat = Mat::zeros(2 * P_max + 1, 2 * angle_max + 1, CV_32S);


	int M = image.rows;
	int N = image.cols;
		
	for (int i = 0; i <M; i++)
		{  for (int j = 0; j <N; j++)
			{     
		    if(image.at<float>(i, j) > 0)//ֻ����Ǳ�����
			    {
				for(int ang = angle_min; ang < angle_max + 1; ang++)
					{  
					int value=(int)(i*cos(ang*3.1415926 / 180.0) + j * sin(ang*3.1415926 / 180.0));
					//cout<<"value:"<<value<<"ang:"<<ang<<"|"<< value + p_len - 1<<"|"<< ang + a_len - 1 <<endl;
					Hough_mat.at<int>(value+p_len-1, ang+a_len-1) = Hough_mat.at<int>(value + p_len-1, ang + a_len-1)+1;
					} 
				}
			}
		}
	return Hough_mat.clone();
}


Mat Fill_Vertical(Mat& g ,int k) 
{
	Mat result_N = Mat::zeros(g.size(), CV_8U);
	int N = g.cols;
	int M = g.rows;

	for (size_t i = 0; i < N; i++)
	{
		for (size_t j = 0; j < M; j++)
		{
			if (j < M - k)
			{
				if (g.at<uchar>(j, i) == 255)
				{
					result_N.at<uchar>(j, i) = 255;
					int k_n = 1;
					while (k_n < k)
					{
						if (g.at<uchar>(j + k_n, i) == 255)
							break;
						else
							k_n = k_n + 1;
					}

					if (k_n > 1 && g.at<uchar>(j + k_n, i) == 255 && k_n < k)
					{
						for (size_t n = 1; n <= k_n; n++)
						{
							result_N.at<uchar>(j + (int)n, i) = 255;
						}
					}
				}
			}
			else {

				result_N.at<uchar>(j, i) = g.at<uchar>(j, i);
			}
		}
	}
	return result_N.clone();
}

void Hough_test() 
{
	Mat Lena = imread("C:/Users/Administrator/Desktop/opencv/air.tif");
	cvtColor(Lena, Lena, COLOR_BGR2GRAY);//ת��Ϊ�Ҷ�ͼ
	image_show(Lena, 1, "ԭͼ");
  
	//һ���ȸ�ͨ�˲�����������˹�任
	f_screem<float>* filter_G = set_f<float>(sf_mode::Gauss25_N);
	Mat GSmat = space_filter_gpu<float, float>("", Lena, filter_G->len, filter_G->postion, filter_G->data, 1);
	//image_show(GSmat, 0.4, "��˹�˲���");

	f_screem<float>* filter_x = set_f<float>(sf_mode::sobel_x_N);
	Mat soble_x = space_filter_gpu<float, float>("", GSmat, filter_x->len, filter_x->postion, filter_x->data, 1);
	image_show(soble_x, 1, "soble_x");

	f_screem<float>* filter_y = set_f<float>(sf_mode::sobel_y_N);
	Mat soble_y = space_filter_gpu<float, float>("", GSmat, filter_y->len, filter_y->postion, filter_y->data, 1);
	image_show(soble_y, 1, "soble_y");


	Mat M_xy;
	sqrt(soble_x.mul(soble_x) + soble_y.mul(soble_y), M_xy);
	///M_xy.convertTo(M_xy, CV_8U);

	double max1, min1;
	cv::Point min_loc1, max_loc1;
	cv::minMaxLoc(M_xy, &min1, &max1, &min_loc1, &max_loc1);
	cout <<"max1:"<< max1<< endl;
	//�����ݶȷ���

	Mat output;
	soble_x.convertTo(soble_x, CV_16SC1);
	soble_y.convertTo(soble_y, CV_16SC1);
	//Canny(Lena,output,max1*0.05,max1*0.15,3,true);
	cv::Canny(soble_x, soble_y, output, max1*0.05, max1*0.15, true);
	image_show(output,1,"canny");

	//��������������ռ�
	Mat H=Hough_gpu(output);
	Mat H_N = H.clone();;
	H_N.convertTo(H_N, CV_8U);
	H_N = H_N * 1.5;
	image_show(H_N,0.3,4, "H_N");

	double max, min;
	cv::Point min_loc, max_loc;
	cv::minMaxLoc(H, &min, &max, &min_loc, &max_loc);

	cout << max << endl;
	cout << "x" << max_loc.x << endl;
	cout << "y" << max_loc.y << endl;


	//ȷ�����ֱ���ϵĵ�
	int M = output.rows;
	int N = output.cols;

	output.convertTo(output,CV_8U);

	Mat result=Mat::zeros(output.size(),CV_8U);

	int angle_min = -90;//����
	int angle_max = 90;

	int P_min = (int)-sqrt(pow(output.rows, 2.0) + pow(output.cols, 2.0));//����
	int P_max = (-1.0)*P_min;

	int p_len = P_max + 1;
	int a_len = angle_max + 1;
	
	int x_bnk= max_loc.x;
	cout << "�Ƕȣ�" << max_loc.x - a_len + 1 << endl;
	while (true) {//�ҵ�90+�����Ԫ��-1
		for (int i = 0; i < M; i++)
		{
			for (int j = 0; j < N; j++)
			{
				if (output.at<uchar>(i, j) > 0 && (int)(i*cos((max_loc.x - a_len + 1) *3.1415926 / 180.0) + j * sin((max_loc.x - a_len + 1)*3.1415926 / 180.0)) == max_loc.y - p_len + 1)//ֻ����Ǳ�����
				//if (output.at<uchar>(i, j) > 0 && (int)(i*cos((max_loc.x - a_len + 1) *3.1415926 / 180.0) + j * sin((max_loc.x - a_len + 1)*3.1415926 / 180.0))>0)//ֻ����Ǳ�����
				{

					result.at <uchar>(i, j) = 255;
				}

			}
		}
		
		H.at<int>(max_loc.y, max_loc.x) = 0;
		cv::minMaxLoc(H, &min, &max, &min_loc, &max_loc);
		//cout << max << endl;
		cout << "x" << max_loc.x << endl;
		cout << "y" << max_loc.y << endl;
		if (x_bnk != max_loc.x || max==0)
			break;
		else
		   x_bnk = max_loc.x;
	}


	////�ҵ�-90+1
	int M_H = H.rows;
	//cout<<"x_bnk:"<< -(x_bnk - a_len + 1)+a_len-1 <<endl;
	x_bnk = -(x_bnk - a_len + 1) + a_len - 1;
	cout<<"x_bnk==:"<< x_bnk <<endl;
	int max_big = 0;
	int max_index = 0;
	for (int i = 0; i < M_H ; i++)
	{
		if (H.at<int>(i, x_bnk) > max_big)
		{
			max_big = H.at<int>(i, x_bnk);
			max_index = i;
		}
	}

	max_loc.x = x_bnk;
	max_loc.y = max_index;
	cout<<"�Ƕȣ�"<< max_loc.x - a_len + 1 <<endl;
	while (true) {//�ҵ�90+�����Ԫ��
		for (int i = 0; i < M; i++)
		{
			for (int j = 0; j < N; j++)
			{
				if (output.at<uchar>(i, j) > 0 && (int)(i*cos((max_loc.x - a_len + 1) *3.1415926 / 180.0) + j * sin((max_loc.x - a_len + 1)*3.1415926 / 180.0)) == max_loc.y - p_len + 1)//ֻ����Ǳ�����
				//if (output.at<uchar>(i, j) > 0 && (int)(i*cos((max_loc.x - a_len + 1) *3.1415926 / 180.0) + j * sin((max_loc.x - a_len + 1)*3.1415926 / 180.0))>0)//ֻ����Ǳ�����
				{

					result.at <uchar>(i, j) = 255;
				}

			}
		}

		H.at<int>(max_loc.y, max_loc.x) = 0;
		cv::minMaxLoc(H, &min, &max, &min_loc, &max_loc);
		cout << max << endl;
		cout << "x" << max_loc.x << endl;
		cout << "y" << max_loc.y << endl;
		if (x_bnk != max_loc.x || max == 0)
			break;
	}
/*
	////90du
	//x_bnk = 180;

	//max_big = 0;
	//max_index = 0;
	//for (int i = 0; i < M_H; i++)
	//{
	//	if (H.at<int>(i, x_bnk) > max_big)
	//	{
	//		max_big = H.at<int>(i, x_bnk);
	//		max_index = i;
	//	}
	//}

	//max_loc.x = x_bnk;
	//max_loc.y = max_index;
	//cout << "�Ƕȣ�" << max_loc.x - a_len + 1 << endl;
	//while (true) {//�ҵ�90+�����Ԫ��
	//	for (int i = 0; i < M; i++)
	//	{
	//		for (int j = 0; j < N; j++)
	//		{
	//			if (output.at<uchar>(i, j) > 0 && (int)(i*cos((max_loc.x - a_len + 1) *3.1415926 / 180.0) + j * sin((max_loc.x - a_len + 1)*3.1415926 / 180.0)) == max_loc.y - p_len + 1)//ֻ����Ǳ�����
	//			//if (output.at<uchar>(i, j) > 0 && (int)(i*cos((max_loc.x - a_len + 1) *3.1415926 / 180.0) + j * sin((max_loc.x - a_len + 1)*3.1415926 / 180.0))>0)//ֻ����Ǳ�����
	//			{

	//				result.at <uchar>(i, j) = 255;
	//			}

	//		}
	//	}


	//	H.at<int>(max_loc.y, max_loc.x) = 0;
	//	cv::minMaxLoc(H, &min, &max, &min_loc, &max_loc);
	//	cout << max << endl;
	//	cout << "x" << max_loc.x << endl;
	//	cout << "y" << max_loc.y << endl;
	//	if (x_bnk != max_loc.x || max == 0)
	//		break;
	//}

	//////90du
	//x_bnk = 0;

	//max_big = 0;
	//max_index = 0;
	//for (int i = 0; i < M_H; i++)
	//{
	//	if (H.at<int>(i, x_bnk) > max_big)
	//	{
	//		max_big = H.at<int>(i, x_bnk);
	//		max_index = i;
	//	}
	//}

	//max_loc.x = x_bnk;
	//max_loc.y = max_index;
	//cout << "�Ƕȣ�" << max_loc.x - a_len + 1 << endl;
	//while (true) {//�ҵ�90+�����Ԫ��
	//	for (int i = 0; i < M; i++)
	//	{
	//		for (int j = 0; j < N; j++)
	//		{
	//			if (output.at<uchar>(i, j) > 0 && (int)(i*cos((max_loc.x - a_len + 1) *3.1415926 / 180.0) + j * sin((max_loc.x - a_len + 1)*3.1415926 / 180.0)) == max_loc.y - p_len + 1)//ֻ����Ǳ�����
	//			//if (output.at<uchar>(i, j) > 0 && (int)(i*cos((max_loc.x - a_len + 1) *3.1415926 / 180.0) + j * sin((max_loc.x - a_len + 1)*3.1415926 / 180.0))>0)//ֻ����Ǳ�����
	//			{

	//				result.at <uchar>(i, j) = 255;
	//			}

	//		}
	//	}

	//	H.at<int>(max_loc.y, max_loc.x) = 0;
	//	cv::minMaxLoc(H, &min, &max, &min_loc, &max_loc);
	//	cout << max << endl;
	//	cout << "x" << max_loc.x << endl;
	//	cout << "y" << max_loc.y << endl;
	//	if (x_bnk != max_loc.x || max == 0)
	//		break;
	//} */ 

	//����result�в�����100�����ط�϶
	image_show(result, 1, "Hough_no_fill2");
	result = Fill_Vertical(result, 150);
	image_show(result, 1, "Hough");
}


//������ֵ����
//cv::Mat getImageofHistogram(const cv::Mat &hist, int zoom);//����ֱ��ͼ
//void Histogram(Mat& image);ͳ��ֱ��ͼ
void show_His(Mat& His_N,char* name,int mode=0) {
	Mat His=His_N.clone();
	His.convertTo(His, CV_8U);
	Histogram(His);
	
	if (mode == 1)
	{
		His.at<float>(0, 0) = 0.0;
	}
  
	His = getImageofHistogram(His, 1);
    image_show(His, 1, name);
}

void Thresholding_test() {

	/*//ͼ1
	//Mat Lena = imread("C:/Users/Administrator/Desktop/opencv/Fig1036a.tif");
	//cvtColor(Lena, Lena, COLOR_BGR2GRAY);//ת��Ϊ�Ҷ�ͼ
	//image_show(Lena, 1, "ԭͼa");

	//Mat Lenaa = imread("C:/Users/Administrator/Desktop/opencv/Fig1036b.tif");
	//cvtColor(Lenaa, Lenaa, COLOR_BGR2GRAY);//ת��Ϊ�Ҷ�ͼ
	//image_show(Lenaa, 1, "ԭͼb");


	//Mat Lenab = imread("C:/Users/Administrator/Desktop/opencv/Fig1036c.tif");
	//cvtColor(Lenab, Lenab, COLOR_BGR2GRAY);//ת��Ϊ�Ҷ�ͼ
	//image_show(Lenab, 1, "ԭͼc");
	//
	//Histogram(Lena);
	//Lena=getImageofHistogram(Lena, 1);
	//image_show(Lena, 1, "aԭͼֱ��ͼ");

	//Histogram(Lenaa);
	//Lenaa = getImageofHistogram(Lenaa, 1);
	//image_show(Lenaa, 1, "bԭͼֱ��ͼ");

	//Histogram(Lenab);
	//Lenab = getImageofHistogram(Lenab, 1);
	//image_show(Lenab, 1, "cԭͼֱ��ͼ");*/

	/*//ͼ2
	//Mat Lenaaa = imread("C:/Users/Administrator/Desktop/opencv/Fig1037aa.tif");
	//cvtColor(Lenaaa, Lenaaa, COLOR_BGR2GRAY);//ת��Ϊ�Ҷ�ͼ
	////image_show(Lena, 1, "ԭͼab");

	//Mat Lenaab = imread("C:/Users/Administrator/Desktop/opencv/Fig1037ab.tif");
	//cvtColor(Lenaab, Lenaab, COLOR_BGR2GRAY);//ת��Ϊ�Ҷ�ͼ
	////image_show(Lenaab, 1, "ԭͼbb");

	//Lenaaa.convertTo(Lenaaa, CV_32F);
	//Lenaab.convertTo(Lenaab, CV_32F);


	//Mat Lenabc = Lenaab+Lenaaa;
	//image_show(Lenabc, 1, "abcԭͼֱ��ͼ");
	//Lenabc.convertTo(Lenabc,CV_8U);

	//Histogram(Lenaab);
	//Lenaab = getImageofHistogram(Lenaab, 1);
	//image_show(Lenaab, 1, "aabԭͼֱ��ͼ");

	//Histogram(Lenabc);
	//Lenabc = getImageofHistogram(Lenabc, 1);
	//image_show(Lenabc, 1, "cbԭͼֱ��ͼ");*/

	//ָ��
	Mat Lena = imread("C:/Users/Administrator/Desktop/opencv/Fig1038a.tif");
	cvtColor(Lena, Lena, COLOR_BGR2GRAY);//ת��Ϊ�Ҷ�ͼ
	image_show(Lena, 1, "ָ��ԭͼ");

	Mat Lena_N=Lena.clone();
	Histogram(Lena);
    Lena=getImageofHistogram(Lena, 1);
    image_show(Lena, 1, "ָ��ֱ��ͼ");
	//����125�ǵ�����������ڵ����Ƚϼ�û��д��ش���
	threshold(Lena_N, Lena_N, 125, 255, 0);
	image_show(Lena_N, 1, "ָ����ֵͼ");

	//Otsu

}

//��Ѱrato������ֵ
int find_Target(Mat& image_N,float rato){
	Mat image = image_N.clone();
	Histogram(image);
	Scalar ss;
	ss=sum(image);

	for(size_t i = 0; i < 256; i++)
	  {
		image.at<float>(i,0)=image.at<float>(i, 0)/ss[0];
	  }

	int result;
	float sum = 0;
	for (size_t i = 0; i < 256; i++)
	{
		if (sum < rato)
		{
			sum = sum + image.at<float>(i, 0);
		}
		else
		{
			result = i;
			break;
		}
	}
	return result;
}

//���԰��Լ������ĸ��ʺ���
Mat Otsu(Mat& image,Mat& probability_N) {
	
	//�����һ��ֱ��ͼ
	Mat Lena=image.clone();
	Mat Lena_N=Lena.clone();
	Histogram(Lena);//row=255
    
	//�������
	Mat probability;
	if (probability_N.empty()){
		Lena.convertTo(probability, CV_32F);
		Scalar ss;
		ss = sum(probability);
		probability = probability / ss[0];
		//cout<< sum(probability)[0] <<endl;
	}
	else {
		probability = probability_N.clone();
		probability.convertTo(probability, CV_32F);
		Scalar ss;
		ss = sum(probability);
		probability = probability / ss[0];
		//cout << sum(probability)[0] << endl;
	}
	//ѡ��һ��Kֵ

	//����ƽ���Ҷ�
	//m1(k)�������ؾ�ֵ
	auto m1_avg = [](Mat hist_pro,int k)->float
	   {Mat hist=hist_pro.clone();
	    hist.convertTo(hist, CV_32F);

		float sum=0;
		float result=0;
		for(int i = 0; i < k; i++)
		   {   result = result + (float)(i)*hist.at<float>(i, 0);
			   sum = hist.at<float>(i, 0) + sum;
		   }
		result = result / sum;
		return result;
	   };

	//m2(k)�������ʾ�ֵ
	auto m2_avg = [](Mat hist_pro, int k)->float
	  { int M = 256;
		Mat hist = hist_pro.clone();
		hist.convertTo(hist, CV_32F);

		float sum = 0;
		float result = 0;
		for (int i = k; i < M; i++)
		{
			result = result + (float)(i)*hist.at<float>(i, 0);
			sum = hist.at<float>(i, 0) + sum;
		}
		result = result / sum;
		return result;
	  };

    //ȫ������ƽ��ֵ
	auto mG_avg = [](Mat hist_pro)->float
	{int M = 256;
	 Mat hist = hist_pro.clone();
	 hist.convertTo(hist, CV_32F);

	 float result = 0;
	 for (int i = 0; i < M; i++)
	   {
		result = result + (float)(i)*hist.at<float>(i, 0);
	   }
	 return result;
	};

	//P1�ۼƸ���
	auto p_sum = [](Mat hist_pro,int k)->float
	{int M = 256;
	 Mat hist = hist_pro.clone();
	 hist.convertTo(hist, CV_32F);

	 float result = 0;
	 for (int i = 0; i < k; i++)
	  {
		result = result + hist.at<float>(i, 0);
	  }
	 return result;
	};

	//ȫ�ַ���
	auto var_G = [](Mat hist_pro)->float
	{int M = 256;
	 Mat hist = hist_pro.clone();
	 hist.convertTo(hist, CV_32F);

	 float m = 0;
	 for(int i = 0; i < M; i++)
	    {
		 m = m + float(i)*hist.at<float>(i, 0);
	     }

	 float result = 0;
	 for(int i = 0; i < M; i++)
	   {
		 result = result + pow((float)i-m,2.0)*hist.at<float>(i, 0);
	    }
	  return result;
	};

	//mk�ۼƸ���
	auto mk_sum = [](Mat hist_pro, int k)->float
	{int M = 256;
	Mat hist = hist_pro.clone();
	hist.convertTo(hist, CV_32F);

	float result = 0;
	for (int i = 0; i < k; i++)
	{
		result = result + (float)(i)*hist.at<float>(i, 0);
	}
	return result;
	};

	//�ֲ�����
	auto var_B = []( float p1, float p2,float mg,float m1 ,float m2, float mk)->float
	{float result;
	 //result = p1 * pow(m1 - mg, 2.0) + p2 * pow(m2 - mg, 2.0);
	 //result = p1*p2*pow((m1-m2),2.0);
	 result = pow(mg*p1 - mk, 2.0)/ (p1*(1 - p1));
	 return result;
	};

	//�������ľֲ�����
	float varG=var_G(probability);//ȫ�ַ���
	float max_var = 0;
	int K=0;
	for(int  k = 0; k < 256; k++)//������󷽲�
	{
		float p1 = p_sum(probability, k);
		float p2 = 1 - p1;

		float mG = mG_avg(probability);
		float m1 = m1_avg(probability, k);
		float m2 = m2_avg(probability, k);

		float mk=mk_sum(probability, k);

		float v_B=var_B(p1,p2,mG,m1,m2,mk);
		if (v_B >= max_var) {
			max_var = v_B;
			K = k;
		}
		//cout<<v_B <<"|"<<k<<endl;
	 }

	//cout<<"���ֵ"<<max_var <<endl;
	//�����n�����ֵ������ƽ��K
	int K_sum = 0;
	int count = 0;
	for (int k = 0; k < 256; k++)//������󷽲�
	{
		float p1 = p_sum(probability, k);
		float p2 = 1 - p1;

		float mG = mG_avg(probability);
		float m1 = m1_avg(probability, k);
		float m2 = m2_avg(probability, k);

		float mk = mk_sum(probability, k);

		float v_B = var_B(p1, p2, mG, m1, m2, mk);
		if (v_B == max_var) {
			K_sum+=k;
			count+=1;
		}
	}

	cout<<"��ֵ��"<< (int)(K_sum/count) <<endl;
	threshold(Lena_N, Lena_N, (int)(K_sum/count), 255, 0);
	//image_show(Lena_N, 1, "�����ֵͼ");
	return  Lena_N.clone();
}


Mat Otsu_two(Mat& image, Mat& probability_N) {

	//�����һ��ֱ��ͼ
	Mat Lena = image.clone();
	Mat Lena_N = Lena.clone();
	Histogram(Lena);//row=255

	//�������
	Mat probability;
	if (probability_N.empty()) {
		Lena.convertTo(probability, CV_32F);
		Scalar ss;
		ss = sum(probability);
		probability = probability / ss[0];
		//cout<<"lena:" <<sum(probability)[0] <<endl;
	}
	else {
		probability = probability_N.clone();
		probability.convertTo(probability, CV_32F);
		Scalar ss;
		ss = sum(probability);
		probability = probability / ss[0];
		//cout <<"probability_N"<<sum(probability)[0] << endl;
	}
	//ѡ��һ��Kֵ

	//����ƽ���Ҷ�
	//m1(k)�������ؾ�ֵ
	auto mk_avg = [](Mat hist_pro, int k1,int k2)->float
	{Mat hist = hist_pro.clone();
	hist.convertTo(hist, CV_32F);

	float sum = 0;
	float result = 0;
	for (int i = k1; i < k2; i++)
	{
		result = result + (float)(i)*hist.at<float>(i, 0);
		sum = hist.at<float>(i, 0) + sum;
	}
	result = result / sum;
	return result;
	};

	//Pk�ۼƸ���
	auto p_sum = [](Mat hist_pro, int k1,int k2)->float
	{int M = 256;
	Mat hist = hist_pro.clone();
	hist.convertTo(hist, CV_32F);

	float result = 0;
	for (int i = k1; i < k2; i++)
	{
		result = result + hist.at<float>(i, 0);
	}
	return result;
	};

	//ȫ������ƽ��ֵ
	auto mG_avg = [](Mat hist_pro)->float
	{int M = 256;
	 Mat hist = hist_pro.clone();
	 hist.convertTo(hist, CV_32F);

	 float result = 0;
	 for (int i = 0; i < M; i++)
	  {
		result = result + (float)(i)*hist.at<float>(i, 0);
	  }
	  return result;
	 };

	//�ֲ�����
	auto var_B = [](float p1, float p2, float p3,float mg, float m1, float m2, float m3)->float
	{float result;
	 result = p1 * pow(m1 - mg, 2.0) + p2 * pow(m2 - mg, 2.0)+p3*pow(m3-mg,2.0);
	 return result;
	 };

	//�������ľֲ�����
	float max_var = 0;
	int K1 = 0;
	int K2 = 0;
	float mg = mG_avg(probability);
	for(int k1 = 0;k1 < 256; k1++)
	{
	  for(int k2 = 0; k2 < 256; k2++)//������󷽲�
		{      
			if (k1<k2) 
			{   //023-63855237
				float p1=p_sum(probability, 0, k1);
				float p2=p_sum(probability, k1, k2);
				float p3=p_sum(probability, k2, 256);
				float m1 = mk_avg(probability,0,k1);
				float m2 = mk_avg(probability, k1, k2);
				float m3 = mk_avg(probability, k2, 256);
			
				float v_B = var_B(p1,p2,p3,mg,m1,m2,m3);
				if(v_B > max_var)
				  {
					max_var = v_B;
					K1 = k1;
					K2 = k2;
					//cout <<p1+p2+p3<<"|"<<v_B << "|" << k1 << "|" << k2 << endl;
				  }
			}
			
		}
	}

	//cout << "��ֵk1��" << (int)K1 << endl;
	//cout << "��ֵk2��" << (int)K2 << endl;

	int K_sum1 = 0;
	int K_sum2 = 0;
	int count = 0;
	for (int k1 = 0; k1 < 256; k1++)
	{
		for (int k2 = 0; k2 < 256; k2++)//������󷽲�
		{
			if (k1 < k2)
			{
				float p1 = p_sum(probability, 0, k1);
				float p2 = p_sum(probability, k1, k2);
				float p3 = p_sum(probability, k2, 256);
				float m1 = mk_avg(probability, 0, k1);
				float m2 = mk_avg(probability, k1, k2);
				float m3 = mk_avg(probability, k2, 256);

				float v_B = var_B(p1, p2, p3, mg, m1, m2, m3);
				if (v_B == max_var)
				{
					K_sum1 += k1;
					K_sum2 += k2;
					count += 1;
				}
			}
			//cout<<v_B <<"|"<<k<<endl;
		}
	}

	//cout<<"���ֵ"<<max_var <<endl;
	//�����n�����ֵ������ƽ��K
	cout << "��ֵk1��" << (int)(K_sum1 / count) << endl;
	cout << "��ֵk2��" << (int)(K_sum2 / count) << endl;

	K1 = (int)(K_sum1 / count);
	K2 = (int)(K_sum2 / count);

	int N = Lena_N.cols;
	int M = Lena_N.rows;
	for (size_t i = 0; i <M; i++)
		  {  for (size_t j = 0; j <N; j++)
			   { 
			    if(Lena_N.at<uchar>(i, j)<=K1)
			       Lena_N.at<uchar>(i, j)=0;
				if(Lena_N.at<uchar>(i, j) >K1 && Lena_N.at<uchar>(i, j)<=K2)
				   Lena_N.at<uchar>(i, j) = 155;
				if(Lena_N.at<uchar>(i, j) >K2)
				   Lena_N.at<uchar>(i, j) = 255;
			   }
		  }
	return  Lena_N.clone();
}

void chapter10()
{
	//1��Hough_test();

    //2\Thresholding_test();

	//3��Otsu()
	//Mat Lena = imread("C:/Users/Administrator/Desktop/opencv/Fig1039a.tif");
	//cvtColor(Lena, Lena, COLOR_BGR2GRAY);//ת��Ϊ�Ҷ�ͼ
	//image_show(Lena, 1, "ԭͼ");
	//Mat Lena_show=Lena.clone();
	//Histogram(Lena_show);
	//Lena_show = getImageofHistogram(Lena_show, 1);
	//image_show(Lena_show, 1, "ֱ��ͼ");
	//Mat pro;
	//Lena=Otsu(Lena,pro);
	//image_show(Lena, 1, "otsuͼ");

	//4������ͼ��ƽ������ȫ����ֵ
	//Mat Lena = imread("C:/Users/Administrator/Desktop/opencv/Fig1040a.tif");//��Ч��ͼ
	//Mat Lena = imread("C:/Users/Administrator/Desktop/opencv/Fig1041a.tif");//��Ч��ͼ
	//cvtColor(Lena, Lena, COLOR_BGR2GRAY);//ת��Ϊ�Ҷ�ͼ
	//image_show(Lena, 1, "ԭͼ");
	//show_His(Lena,"5*5�˲�ǰ-ֱ��ͼ");
	//Mat pro;
	//Mat Lena_show=Otsu(Lena,pro);
	//image_show(Lena_show, 1, "5*5�˲�ǰ-Otsuͼ");

	//f_screem<float>* filter_G = set_f<float>(sf_mode::avg_5);
	//Mat avg_mat = space_filter_gpu<float, float>("", Lena, filter_G->len, filter_G->postion, filter_G->data, 1);
	//image_show(avg_mat, 1, "5*5�˲���ͼ");
	//avg_mat.convertTo(avg_mat, CV_8U);
	//show_His(avg_mat, "5*5�˲���-ֱ��ͼ");
	//pro;
	//Lena_show = Otsu(avg_mat,pro);
	//image_show(Lena_show, 1, "5*5�˲���-Otsuͼ");

	//5�����ñ�Ե�Ľ�ȫ����ֵ����
	//1)�ݶ�Ѱ�ұ߽�
	//Mat lena = imread("c:/users/administrator/desktop/opencv/fig1041a.tif");//��Ч��ͼ
	//cvtColor(lena, lena, COLOR_BGR2GRAY);//ת��Ϊ�Ҷ�ͼ
	//Mat lena_o=lena.clone();
	//Mat lena_out = lena.clone();
	//image_show(lena, 1, "ԭͼ");
	//show_His(lena,"�˻�ǰ-ֱ��ͼ",0);
	//lena=sobel_grad(lena,1);
	//lena.convertTo(lena, CV_8U);
	//
	//cout<<"��ȡ��ֵ"<<find_Target(lena, 0.997) <<endl;
 //   threshold(lena, lena, find_Target(lena, 0.997), 1, 0);
	//image_show(lena, 1, "�ݶ�ͼ");

	//lena_o=lena.mul(lena_o);
	//image_show(lena_o, 1, "�˻���ͼ");
	//show_His(lena_o, "�˻���-ֱ��ͼ", 1);

	//Histogram(lena_o);
	//lena_o.at<float>(0, 0) = 0;
	//lena_out =Otsu(lena_out, lena_o);
	//image_show(lena_out, 1, "���ͼ");

	//
	//2)��������˹Ѱ�ұ߽� 	Laplace8_N = 6,
	//Mat lena = imread("c:/users/administrator/desktop/opencv/fig1043a.tif");//��Ч��ͼ
	//cvtColor(lena, lena, COLOR_BGR2GRAY);//ת��Ϊ�Ҷ�ͼ
	//Mat lena_o = lena.clone();
	//image_show(lena, 1, "ԭͼ");
	//show_His(lena, "�˻�ǰ-ֱ��ͼ", 0);

	//Mat pro;
	//Mat Lena_show=Otsu(lena,pro);
	//image_show(Lena_show, 1, "ֱ��otsuԭͼ");

	//////������˹
	//f_screem<float>* filter_G = set_f<float>(sf_mode::Laplace8_N);
	//Mat laplace_mat = space_filter_gpu<float, float>("", lena, filter_G->len, filter_G->postion, filter_G->data, 1);
	//laplace_mat = abs(laplace_mat);
	//laplace_mat.convertTo(laplace_mat, CV_8U);
	//image_show(laplace_mat, 1, "������˹");
	//
	////99.7%���ݶ�ֵ
	//cout<<"������˹�ݶ���ֵ:"<<find_Target(laplace_mat,0.995)<<endl;

	//threshold(laplace_mat, laplace_mat, find_Target(laplace_mat, 0.995),1, 0);
	//image_show(laplace_mat, 1, "��ֵ�����Ժ��������˹");

	//lena =lena.mul(laplace_mat);
	//image_show(lena, 1, "�˻���ͼ");
	//show_His(lena, "�˻���-ֱ��ͼ", 1);
	//
	//Histogram(lena);
	//lena.at<float>(0, 0) = 0;
	//lena_o =Otsu(lena_o, lena);
	//image_show(lena_o, 1, "���ͼ");

	//6������ֵ����
	Mat lena = imread("c:/users/administrator/desktop/opencv/Fig1045a.tif");//��Ч��ͼ
	cvtColor(lena, lena, COLOR_BGR2GRAY);//ת��Ϊ�Ҷ�ͼ
	Mat lena_o = lena.clone();
	image_show(lena, 1, "ԭͼ");
	show_His(lena, "�˻�ǰ-ֱ��ͼ", 0);

	Mat pro;
    Mat image_two=Otsu_two(lena, pro);
    image_show(image_two, 1, "2��ֵotsuͼ");
	//7���ɱ���ֵ����
   
}