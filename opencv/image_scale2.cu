#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <opencv2/opencv.hpp>
#include <iostream>
#include <string>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "math.h"
#include <>
#include <sstream>
#include <hiprand.h>

using namespace std;
using namespace cv;

//����CUDA����
texture <uchar, hipTextureType2D, hipReadModeElementType> refTex_double_linear;//���ڼ���˫���Բ�ֵ
texture <uchar, hipTextureType2D, hipReadModeNormalizedFloat> refTex_double_linear_cuda;//���ڼ���cuda�����Դ�˫���Բ�ֵ
texture <uchar, hipTextureType2D, hipReadModeElementType> refTex_neareast_point;//���ڼ������point��ֵ
texture <uchar, hipTextureType2D, hipReadModeElementType> refTex_Bicubic;//���ڼ���˫�������Բ�ֵ

//����CUDA����
hipArray* cuArray_double_linear;;//���ڼ���˫���Բ�ֵ
hipArray* cuArray_double_linea_cuda;//���ڼ���cuda�����Դ�˫���Բ�ֵ
hipArray* cuArray_neareast_point;//���ڼ������point��ֵ
hipArray* cuArray_Bicubic;//���ڼ���˫�������Բ�ֵ

//ͨ����
hipChannelFormatDesc cuDesc = hipCreateChannelDesc<uchar>();

//һ��˫���Բ�ֵ����
//��1��˫���Բ�ֵ                     //cuda����x��y�����Ƕ��������Ͻǵ���������ϵ
//                                        ��0��0��----x------>
//(x1,y2)  (x2,y2)
//Q12---------Q22                         |
//     (x,y)                              y
//Q11---------Q21                         |
//(x1,y1) ��x2,y1��

//ע��Q11��Q12��Q22��Q21Ϊ��ӽ��ı���ֵ�㣨x_des,y_des) �ĸ����ص�ֵ����Χ�ڡ�0-255����
//x1,x2,y1,y2Ϊ���ص�Q11��Q12��Q22��Q21���������꣬�Լ�д��˫���Բ�ֵ
__device__ uchar interpolation(int x1, int y2, float x_des, float y_des) {
	int x2 = x1 + 1;
	int y1 = y2 + 1;
	uchar rezult = 0;

	uchar fQ11 = tex2D(refTex_double_linear, x1, y1);
	uchar fQ12 = tex2D(refTex_double_linear, x1, y2);
	uchar fQ22 = tex2D(refTex_double_linear, x2, y2);
	uchar fQ21 = tex2D(refTex_double_linear, x2, y1);

	rezult = (uchar)floor((((float)fQ11 / (x2 - x1) * (y2 - y1)) * (x2 - x_des) * (y2 - y_des) + ((float)fQ21 / (x2 - x1) * (y2 - y1)) * (x_des - x1) * (y2 - y_des)
		+ ((float)fQ12 / (x2 - x1) * (y2 - y1)) * (x2 - x_des) * (y_des - y1) + ((float)fQ22 / (x2 - x1) * (y2 - y1)) * (x_des - x1) * (y_des - y1)));

	return rezult;
}

//˫���Բ�ֵ
//��ԭͼ���в�ֵ���������Ժ��ͼ��,imgHeight_des_dԭͼ��, imgWidth_des_dԭͼ��,imgh_rato_d ���ų��ȱ���, imgw_rato_d���ſ�ȱ���
__global__ void weightAddKerke_double_linear(uchar* pDstImgData, int imgHeight_des_d, int imgWidth_des_d, float imgh_rato_d, float imgw_rato_d)
{   //printf("threadIdx,x=%d",threadIdx.x);
	const int tidx = blockDim.x * blockIdx.x + threadIdx.x;
	const int tidy = blockDim.y * blockIdx.y + threadIdx.y;
	if (tidx < imgWidth_des_d && tidy < imgHeight_des_d)
	{
		int idx = tidy * imgWidth_des_d + tidx;
		float x_des = (tidx + 0.5) / imgw_rato_d - 0.5;
		float y_des = (tidy + 0.5) / imgh_rato_d - 0.5;
		int x1 = (int)floor(x_des); //ȡ�ĸ���ӽ�Ԫ���У����Ͻǵ�Ԫ��
		int y2 = (int)floor(y_des);
		pDstImgData[idx] = interpolation(x1, y2, x_des, y_des);
		//printf("value=%u,%d,%d,%f,%f \n", pDstImgData[idx], x1, y2, x_des, y_des);
	}
}

//cuda���� �Դ���tex2D˫���Բ�ֵ��ֵ����Ҫʹ��float����
//��ԭͼ���в�ֵ���������Ժ��ͼ��,imgHeight_des_dԭͼ��, imgWidth_des_dԭͼ��,imgh_rato_d ���ų��ȱ���, imgw_rato_d���ſ�ȱ���
__global__ void weightAddKerkel_double_linear_cuda(uchar* pDstImgData, int imgHeight_des_d, int imgWidth_des_d, float imgh_rato_d, float imgw_rato_d)
{
	const int tidx = blockDim.x * blockIdx.x + threadIdx.x;
	const int tidy = blockDim.y * blockIdx.y + threadIdx.y;

	if (tidx < imgWidth_des_d && tidy < imgHeight_des_d)
	{
		int idx = tidy * imgWidth_des_d + tidx;
		float x_des = (tidx + 0.5) / imgw_rato_d - 0.5;
		float y_des = (tidy + 0.5) / imgh_rato_d - 0.5;
		pDstImgData[idx] = tex2D(refTex_double_linear_cuda, x_des + 0.5, y_des + 0.5) * 255;
		//printf("value=%u,%f,%f \n", pDstImgData[idx], x_des, y_des);
	}
}


//������������Բ�ֵ
//��ԭͼ���в�ֵ���������Ժ��ͼ��,imgHeight_des_dԭͼ��, imgWidth_des_dԭͼ��,imgh_rato_d ���ų��ȱ���, imgw_rato_d���ſ�ȱ���
__global__ void weightAddKerke_neareast_point(uchar* pDstImgData, int imgHeight_des_d, int imgWidth_des_d, float imgh_rato_d, float imgw_rato_d)
{   //printf("threadIdx,x=%d",threadIdx.x);
	const int tidx = blockDim.x * blockIdx.x + threadIdx.x;
	const int tidy = blockDim.y * blockIdx.y + threadIdx.y;
	if (tidx < imgWidth_des_d && tidy < imgHeight_des_d)
	{
		int idx = tidy * imgWidth_des_d + tidx;
		float x_des = (tidx) / imgw_rato_d;
		float y_des = (tidy) / imgh_rato_d;
		int x1 = (int)floor(x_des); //ȡ�ĸ���ӽ�Ԫ���У����Ͻǵ�Ԫ��
		int y2 = (int)floor(y_des);
		pDstImgData[idx] = tex2D(refTex_neareast_point, x1, y2);
		//printf("value=%u,%d,%d,%f,%f \n", pDstImgData[idx], x1, y2, x_des, y_des);
	}
}


//����˫���β�ֵBicubi��ֵ����
__device__ float Bicubic(float a, float x) {
	float abs_x = abs(x);
	if (abs_x<= 1.0)
    {
		return (a+2)*pow(abs_x, 3) - (a + 3)*pow(abs_x, 2) + 1;
	}
	else if(1<abs(x)<2)
	{
		return a * pow(abs_x, 3) - 5 * a*pow(abs_x, 2) + 8 * a*abs_x - 4 * a;
	}
	else { 
		return 0.0f;
	}
}

//˫��������ֵ
__global__ void weightAddKerkel_Bicubic(uchar* pDstImgData, int imgHeight_des_d, int imgWidth_des_d, float imgh_rato_d, float imgw_rato_d)
{
	const int tidx = blockDim.x * blockIdx.x + threadIdx.x;
	const int tidy = blockDim.y * blockIdx.y + threadIdx.y;

	if (tidx < imgWidth_des_d && tidy < imgHeight_des_d)
	{
		float a = -0.5;

		int idx = tidy * imgWidth_des_d + tidx;
		float x_des = (tidx + 0.5) / imgw_rato_d - 0.5;
		float y_des = (tidy + 0.5) / imgh_rato_d - 0.5;
	    int x = (int)floor(x_des); //ȡ�ĸ���ӽ�Ԫ���У����Ͻǵ�Ԫ��
		int y = (int)floor(y_des);
		float u = x_des - x;
		float v = y_des - y;
		//�ҵ�16����
		float k_i0 = Bicubic(a,1.0+u);
		float k_i1 = Bicubic(a, u);
		float k_i2 = Bicubic(a, 1.0-u);
		float k_i3 = Bicubic(a, 2.0-u);
		float k_j0 = Bicubic(a, 1.0+v);
		float k_j1 = Bicubic(a, v);
		float k_j2 = Bicubic(a, 1.0-v);
		float k_j3 = Bicubic(a, 2.0-v);

		pDstImgData[idx] = tex2D(refTex_Bicubic,x - 1, y - 1)*k_i0*k_j0 + tex2D(refTex_Bicubic,x, y - 1)*k_i1*k_j0 + tex2D(refTex_Bicubic,x + 1, y - 1)*k_i2*k_j0 + tex2D(refTex_Bicubic,x + 2, y - 1)*k_i3*k_j0 +
			tex2D(refTex_Bicubic,x - 1, y)*k_i0*k_j1 + tex2D(refTex_Bicubic,x, y)*k_i1*k_j1 + tex2D(refTex_Bicubic,x + 1, y)*k_i2*k_j1 + tex2D(refTex_Bicubic,x + 2, y)*k_i3*k_j1 +
			tex2D(refTex_Bicubic,x - 1, y +1)*k_i0*k_j2 + tex2D(refTex_Bicubic,x, y+1)*k_i1*k_j2 + tex2D(refTex_Bicubic,x + 1, y+1)*k_i2*k_j2 + tex2D(refTex_Bicubic,x + 2, y+1)*k_i3*k_j2 +
			tex2D(refTex_Bicubic,x - 1, y +2)*k_i0*k_j3 + tex2D(refTex_Bicubic,x, y+2)*k_i1*k_j3 + tex2D(refTex_Bicubic,x + 1, y+2)*k_i2*k_j3 + tex2D(refTex_Bicubic,x + 2, y+2)*k_i3*k_j3;
		    //printf("value=%u,%f,%f \n", pDstImgData[idx], x_des, y_des);
	}
}

//mode=0,�����ֵ��ʽ��1=˫���Բ�ֵ��ʽ��2=˫���β�ֵ��ʽ
void image_zooming(char* path = "C:/Users/Administrator/Desktop/lena.jpg",float x_rato=2.0,float y_rato=2.0,int mode=0) 
{   

	float x_rato_less = x_rato;
	float y_rato_less = y_rato;

	if (mode == 0)//�����ֵ��ʽ
	{
		Mat Lena = imread("C:/Users/Administrator/Desktop/lena.jpg");
		cvtColor(Lena, Lena, COLOR_BGR2GRAY);//ת��Ϊ�Ҷ�ͼ
	    
		int imgWidth_src = Lena.cols;//ԭͼ���
		int imgHeight_src = Lena.rows;//ԭͼ���
		int channels = Lena.channels();

		int imgWidth_des_less = floor(imgWidth_src * x_rato_less);//��Сͼ��
		int imgHeight_des_less = floor(imgHeight_src * y_rato_less);//��Сͼ��

	   //����1��������
		hipError_t t;
		refTex_neareast_point.addressMode[0] = hipAddressModeClamp;
		refTex_neareast_point.addressMode[1] = hipAddressModeClamp;
		refTex_neareast_point.normalized = false;
		refTex_neareast_point.filterMode = hipFilterModePoint;
		//��cuArray������
		hipMallocArray(&cuArray_neareast_point, &cuDesc, imgWidth_src, imgHeight_src);
		t = hipBindTextureToArray(refTex_neareast_point, cuArray_neareast_point);
		//�������ݵ�hipArray
		t = hipMemcpyToArray(cuArray_neareast_point, 0, 0, Lena.data, imgWidth_src * imgHeight_src * sizeof(uchar), hipMemcpyHostToDevice);

		//��������Ժ���cpu��ͼ��
		Mat dstImg1 = Mat::zeros(imgHeight_des_less, imgWidth_des_less, CV_8UC1);//��С

		//��������Ժ���cuda�ϵ�ͼ��
		uchar* pDstImgData1 = NULL;
		t = hipMalloc(&pDstImgData1, imgHeight_des_less * imgWidth_des_less * sizeof(uchar));

		dim3 block(8, 8);
		dim3 grid((imgWidth_des_less + block.x - 1) / block.x, (imgHeight_des_less + block.y - 1) / block.y);

		weightAddKerke_neareast_point<< <grid, block >> > (pDstImgData1, imgHeight_des_less, imgWidth_des_less, y_rato_less, x_rato_less);
		hipDeviceSynchronize();

		//��GPU����������ݵ�CPU
		t = hipMemcpy(dstImg1.data, pDstImgData1, imgWidth_des_less * imgHeight_des_less * sizeof(uchar)*channels, hipMemcpyDeviceToHost);
		hipFree(cuArray_neareast_point);
		hipFree(pDstImgData1);
		//namedWindow("cuda_point�����ֵ��", WINDOW_NORMAL);
		imshow("cuda_point�����ֵ��", dstImg1);
		imwrite("C:/Users/Administrator/Desktop/ͼƬ/Gray_Image0.jpg", dstImg1);
		/*waitKey(0);*/
	}

	if (mode == 1) {
		Mat Lena = imread("C:/Users/Administrator/Desktop/lena.jpg");
		cvtColor(Lena, Lena, COLOR_BGR2GRAY);//ת��Ϊ�Ҷ�ͼ

		int imgWidth_src = Lena.cols;//ԭͼ���
		int imgHeight_src = Lena.rows;//ԭͼ���
		int channels = Lena.channels();

		int imgWidth_des_less = floor(imgWidth_src * x_rato_less);//��Сͼ��
		int imgHeight_des_less = floor(imgHeight_src * y_rato_less);//��Сͼ��

	   //����1��������
		hipError_t t;
		refTex_double_linear.addressMode[0] = hipAddressModeClamp;
		refTex_double_linear.addressMode[1] = hipAddressModeClamp;
		refTex_double_linear.normalized = false;
		refTex_double_linear.filterMode = hipFilterModePoint;
		//��cuArray������
		hipMallocArray(&cuArray_double_linear, &cuDesc, imgWidth_src, imgHeight_src);
		t = hipBindTextureToArray(refTex_double_linear, cuArray_double_linear);
		//�������ݵ�hipArray
		t = hipMemcpyToArray(cuArray_double_linear, 0, 0, Lena.data, imgWidth_src * imgHeight_src * sizeof(uchar), hipMemcpyHostToDevice);

		//��������Ժ���cpu��ͼ��
		Mat dstImg1 = Mat::zeros(imgHeight_des_less, imgWidth_des_less, CV_8UC1);//��С

		//��������Ժ���cuda�ϵ�ͼ��
		uchar* pDstImgData1 = NULL;
		t = hipMalloc(&pDstImgData1, imgHeight_des_less * imgWidth_des_less * sizeof(uchar));

		dim3 block(8, 8);
		dim3 grid((imgWidth_des_less + block.x - 1) / block.x, (imgHeight_des_less + block.y - 1) / block.y);

		weightAddKerke_double_linear << <grid, block >> > (pDstImgData1, imgHeight_des_less, imgWidth_des_less, y_rato_less, x_rato_less);
		hipDeviceSynchronize();

		//��GPU����������ݵ�CPU
		t = hipMemcpy(dstImg1.data, pDstImgData1, imgWidth_des_less * imgHeight_des_less * sizeof(uchar)*channels, hipMemcpyDeviceToHost);
		hipFree(cuArray_double_linear);
		hipFree(pDstImgData1);
		//namedWindow("�Լ���д˫���Բ�ֵ��", WINDOW_NORMAL);
		imshow("�Լ���д˫���Բ�ֵ��", dstImg1);
		imwrite("C:/Users/Administrator/Desktop/ͼƬ/Gray_Image1.jpg", dstImg1);
		//waitKey(0);
	}

	if (mode == 2) {//˫���β�ֵ
		Mat Lena = imread("C:/Users/Administrator/Desktop/lena.jpg");
		cvtColor(Lena, Lena, COLOR_BGR2GRAY);//ת��Ϊ�Ҷ�ͼ

		int imgWidth_src = Lena.cols;//ԭͼ���
		int imgHeight_src = Lena.rows;//ԭͼ���
		int channels = Lena.channels();

		int imgWidth_des_less = floor(imgWidth_src * x_rato_less);//��Сͼ��
		int imgHeight_des_less = floor(imgHeight_src * y_rato_less);//��Сͼ��

	   //����1��������
		hipError_t t;
		refTex_Bicubic.addressMode[0] = hipAddressModeClamp;
		refTex_Bicubic.addressMode[1] = hipAddressModeClamp;
		refTex_Bicubic.normalized = false;
		refTex_Bicubic.filterMode = hipFilterModePoint;
		//��cuArray������
		hipMallocArray(&cuArray_Bicubic, &cuDesc, imgWidth_src, imgHeight_src);
		t = hipBindTextureToArray(refTex_Bicubic, cuArray_Bicubic);
		//�������ݵ�hipArray
		t = hipMemcpyToArray(cuArray_Bicubic, 0, 0, Lena.data, imgWidth_src * imgHeight_src * sizeof(uchar), hipMemcpyHostToDevice);

		//��������Ժ���cpu��ͼ��
		Mat dstImg1 = Mat::zeros(imgHeight_des_less, imgWidth_des_less, CV_8UC1);//��С

		//��������Ժ���cuda�ϵ�ͼ��
		uchar* pDstImgData1 = NULL;
		t = hipMalloc(&pDstImgData1, imgHeight_des_less * imgWidth_des_less * sizeof(uchar));

		dim3 block(8, 8);
		dim3 grid((imgWidth_des_less + block.x - 1) / block.x, (imgHeight_des_less + block.y - 1) / block.y);

		weightAddKerkel_Bicubic << <grid, block >> > (pDstImgData1, imgHeight_des_less, imgWidth_des_less, y_rato_less, x_rato_less);
		hipDeviceSynchronize();

		//��GPU����������ݵ�CPU
		t = hipMemcpy(dstImg1.data, pDstImgData1, imgWidth_des_less * imgHeight_des_less * sizeof(uchar)*channels, hipMemcpyDeviceToHost);
		hipFree(cuArray_Bicubic);
		hipFree(pDstImgData1);
		//namedWindow("˫�����Բ�ֵ��", WINDOW_NORMAL);
		imshow("˫�����Բ�ֵ��", dstImg1);
		imwrite("C:/Users/Administrator/Desktop/ͼƬ/Gray_Image2.jpg", dstImg1);
		//waitKey(0);
	}

	if (mode == 3) {//cuda�����Դ�˫���Բ�ֵ
		Mat Lena = imread("C:/Users/Administrator/Desktop/lena.jpg");
		cvtColor(Lena, Lena, COLOR_BGR2BGRA);//
	    cvtColor(Lena, Lena, COLOR_BGRA2GRAY);// 


		int imgWidth_src = Lena.cols;//ԭͼ���
		int imgHeight_src = Lena.rows;//ԭͼ���
		int channels = Lena.channels();

		int imgWidth_des_less = floor(imgWidth_src * x_rato_less);//��Сͼ��
		int imgHeight_des_less = floor(imgHeight_src * y_rato_less);//��Сͼ��

	   //����1��������
		hipError_t t;
		refTex_double_linear_cuda.addressMode[0] = hipAddressModeClamp;
		refTex_double_linear_cuda.addressMode[1] = hipAddressModeClamp;
		refTex_double_linear_cuda.normalized = false;
		refTex_double_linear_cuda.filterMode = hipFilterModeLinear;
		//��cuArray������
		hipMallocArray(&cuArray_double_linea_cuda, &cuDesc, imgWidth_src, imgHeight_src);
		t = hipBindTextureToArray(refTex_double_linear_cuda, cuArray_double_linea_cuda);
		//�������ݵ�hipArray
		t = hipMemcpyToArray(cuArray_double_linea_cuda, 0, 0, Lena.data, imgWidth_src * imgHeight_src * sizeof(uchar), hipMemcpyHostToDevice);

		//��������Ժ���cpu��ͼ��
		Mat dstImg1 = Mat::zeros(imgHeight_des_less, imgWidth_des_less, CV_8UC1);//��С

		//��������Ժ���cuda�ϵ�ͼ��
		uchar* pDstImgData1 = NULL;
		t = hipMalloc(&pDstImgData1, imgHeight_des_less * imgWidth_des_less * sizeof(uchar));

		dim3 block(8, 8);
		dim3 grid((imgWidth_des_less + block.x - 1) / block.x, (imgHeight_des_less + block.y - 1) / block.y);

		weightAddKerkel_double_linear_cuda << <grid, block >> > (pDstImgData1, imgHeight_des_less, imgWidth_des_less, y_rato_less, x_rato_less);
		hipDeviceSynchronize();

		//��GPU����������ݵ�CPU
		t = hipMemcpy(dstImg1.data, pDstImgData1, imgWidth_des_less * imgHeight_des_less * sizeof(uchar)*channels, hipMemcpyDeviceToHost);
		hipFree(cuArray_double_linea_cuda);
		hipFree(pDstImgData1);
		namedWindow("cuda�Դ�˫���Բ�ֵ��", WINDOW_NORMAL);
		imshow("cuda�Դ�˫���Բ�ֵ��", dstImg1);
		imwrite("C:/Users/Administrator/Desktop/ͼƬ/Gray_Image3.jpg", dstImg1);
	/*	waitKey(0);*/
	}

}


//�ġ��ڲ�ֵ��-��ɾ���㷨��Row and column deletion)= ������ֵ�������ߵȼ�
texture <uchar, hipTextureType2D, hipReadModeElementType> refTex_RCDeletion;//���ڼ���RCDeletion
hipArray* cuArray_RCDeletion;//���ڼ���RCDeletion
//�㷨�򵥽��ܣ�1��opencv��ͼ�����ؼ��ٵ�ż������2���á���ͷ�����㷨��ԭͼ��Ѱ������
//��·��ʱ�������ļ�·������mat��û·��ʱ������mat
Mat RCDeletion(const char* path ,Mat lena_o, float x_rato = 2.0, float y_rato = 2.0,int ifhd=0) 
{
	Mat Lena;
	if (strlen(path) == 0) {
	   Lena=lena_o.clone();
	}
	else {
		Lena = imread(path);
	}

	if (ifhd == 0)//���ǻҶ�ͼҪ����ת��
		cvtColor(Lena, Lena, COLOR_BGR2GRAY);//ת��Ϊ�Ҷ�ͼ

	float x_rato_less = x_rato;
	float y_rato_less = y_rato;

	int imgWidth_src = Lena.cols;//ԭͼ���
	int imgHeight_src = Lena.rows;//ԭͼ���
	int channels = Lena.channels();

	int imgWidth_des_less = floor(imgWidth_src * x_rato_less);//��Сͼ��
	int imgHeight_des_less = floor(imgHeight_src * y_rato_less);//��Сͼ��

   //����1��������
	hipError_t t;
	refTex_RCDeletion.addressMode[0] = hipAddressModeClamp;
	refTex_RCDeletion.addressMode[1] = hipAddressModeClamp;
	refTex_RCDeletion.normalized = false;
	refTex_RCDeletion.filterMode = hipFilterModePoint;
	//��cuArray������
	hipMallocArray(&cuArray_RCDeletion, &cuDesc, imgWidth_src, imgHeight_src);
	t = hipBindTextureToArray(refTex_neareast_point, cuArray_RCDeletion);
	//�������ݵ�hipArray
	t = hipMemcpyToArray(cuArray_RCDeletion, 0, 0, Lena.data, imgWidth_src * imgHeight_src * sizeof(uchar), hipMemcpyHostToDevice);

	//��������Ժ���cpu��ͼ��
	Mat dstImg1 = Mat::zeros(imgHeight_des_less, imgWidth_des_less, CV_8UC1);//��С

	//��������Ժ���cuda�ϵ�ͼ��
	uchar* pDstImgData1 = NULL;
	t = hipMalloc(&pDstImgData1, imgHeight_des_less * imgWidth_des_less * sizeof(uchar));

	dim3 block(8, 8);
	dim3 grid((imgWidth_des_less + block.x - 1) / block.x, (imgHeight_des_less + block.y - 1) / block.y);

	weightAddKerke_neareast_point << <grid, block >> > (pDstImgData1, imgHeight_des_less, imgWidth_des_less, y_rato_less, x_rato_less);
	hipDeviceSynchronize();

	//��GPU����������ݵ�CPU
	t = hipMemcpy(dstImg1.data, pDstImgData1, imgWidth_des_less * imgHeight_des_less * sizeof(uchar)*channels, hipMemcpyDeviceToHost);
	hipFree(cuArray_neareast_point);
	hipFree(pDstImgData1);
	
	//namedWindow("cuda_point�����ֵ��", WINDOW_NORMAL);
	//imshow("cuda_point�����ֵ��", dstImg1);
	return dstImg1.clone();
	//imwrite("C:/Users/Administrator/Desktop/ͼƬ/Gray_Image0.jpg", dstImg1);
	/*waitKey(0);*/
}

//�塢ƽ������ƽ�������˲�����

//��·��ʱ�������ļ�·������mat��û·��ʱ������mat
texture <uchar, hipTextureType2D, hipReadModeElementType> refTex_filter_N;//���ڼ���RCDeletion
hipArray* cuArray_filter_N;//���ڼ���RCDeletion

//��ԭͼ���в�ֵ���������Ժ��ͼ��,imgHeight_des_dԭͼ��, imgWidth_des_dԭͼ��,imgh_rato_d ���ų��ȱ���, imgw_rato_d���ſ�ȱ���
__global__ void weightAddKerke_filter_N(uchar* pDstImgData, int imgHeight_des_d, int imgWidth_des_d, float imgh_rato_d, float imgw_rato_d,int* filter_cute,float divide)
{   //printf("threadIdx,x=%d",threadIdx.x);
	const int tidx = blockDim.x * blockIdx.x + threadIdx.x;
	const int tidy = blockDim.y * blockIdx.y + threadIdx.y;
	if (tidx < imgWidth_des_d && tidy < imgHeight_des_d)
	{
		int idx = tidy * imgWidth_des_d + tidx;
		float x_des = (tidx + 0.5) / imgw_rato_d - 0.5;
		float y_des = (tidy + 0.5) / imgh_rato_d - 0.5;
		int x = (int)floor(x_des); //ȡ�ĸ���ӽ�Ԫ���У����Ͻǵ�Ԫ��
		int y = (int)floor(y_des);
		pDstImgData[idx] = (uchar)((tex2D(refTex_filter_N, x-1, y-1)*filter_cute[0] + tex2D(refTex_filter_N, x, y-1)*filter_cute[1] + tex2D(refTex_filter_N, x+1, y-1)*filter_cute[2]
			+ tex2D(refTex_filter_N, x-1, y)*filter_cute[3] + tex2D(refTex_filter_N, x, y)*filter_cute[4] + tex2D(refTex_filter_N, x+1, y)*filter_cute[5]
			+ tex2D(refTex_filter_N, x - 1, y + 1)*filter_cute[6] + tex2D(refTex_filter_N, x, y + 1)*filter_cute[7] + tex2D(refTex_filter_N, x + 1, y + 1)*filter_cute[8]) /divide);
		//printf("value=%u,%d,%d,%f,%f \n", pDstImgData[idx], x1, y2, x_des, y_des);
	}
}

//ifhd �ж��Ƿ������ͼ��Ϊһ���Ҷ�ͼ���Ǿ�����ת��
Mat filter_N(const char* path, Mat lena_o,int mode=0,int ifhd=0) {
	int* filter_cute;
	hipMallocManaged((void**)&filter_cute, 3 * 3 * sizeof(uchar));//ͳһ��ַ����Ҫ�Կ�֧��ͳһ��ַ
	float divide = 0.0;

	if (mode == 0) {
	    //mode=0 ƽ���˲���
	    //[[1��1��1],
	    //[1, 1, 1],
        //[1, 1, 1]]
		for (int i = 0; i < 9; i++)
		    {filter_cute[i] = 1;}
		divide = 9.0f;
	}
	else {
		//mode=1
	    //[[1��2��1]
        //[2, 4, 2]
        //[1, 2, 1]]
		filter_cute[0] = 1;   //����ط�����Ƕ���һ����
		filter_cute[1] = 2;
		filter_cute[2] = 1;
		filter_cute[3] = 2;
		filter_cute[4] = 4;
		filter_cute[5] = 2;
		filter_cute[6] = 1;
		filter_cute[7] = 2;
		filter_cute[8] = 1;
		divide = 16.0f;
	}

	Mat Lena;
	if (strlen(path) == 0) {
		Lena = lena_o.clone();
	}
	else {
		Lena = imread(path);
	}

	if(ifhd==0)//���ǻҶ�ͼҪ����ת��
	  cvtColor(Lena, Lena, COLOR_BGR2GRAY);//ת��Ϊ�Ҷ�ͼ
	  
	float x_rato_less = 1;//����ͼ�����ز���
	float y_rato_less = 1;//Ϊ�˼��ٴ���Ķ�����ʵӦ�ü򻯵�

	int imgWidth_src = Lena.cols;//ԭͼ���
	int imgHeight_src = Lena.rows;//ԭͼ���
	int channels = Lena.channels();

	int imgWidth_des_less = floor(imgWidth_src * x_rato_less);//��Сͼ��
	int imgHeight_des_less = floor(imgHeight_src * y_rato_less);//��Сͼ��

   //����1��������
	hipError_t t;
	refTex_filter_N.addressMode[0] = hipAddressModeClamp;
	refTex_filter_N.addressMode[1] = hipAddressModeClamp;
	refTex_filter_N.normalized = false;
	refTex_filter_N.filterMode = hipFilterModePoint;
	
	//��cuArray������
	hipMallocArray(&cuArray_filter_N, &cuDesc, imgWidth_src, imgHeight_src);
	t = hipBindTextureToArray(refTex_filter_N, cuArray_filter_N);
	//�������ݵ�hipArray
	t = hipMemcpyToArray(cuArray_filter_N, 0, 0, Lena.data, imgWidth_src * imgHeight_src * sizeof(uchar), hipMemcpyHostToDevice);

	//��������Ժ���cpu��ͼ��
	Mat dstImg1 = Mat::zeros(imgHeight_des_less, imgWidth_des_less, CV_8UC1);//��С

	//��������Ժ���cuda�ϵ�ͼ��
	uchar* pDstImgData1 = NULL;
	t = hipMalloc(&pDstImgData1, imgHeight_des_less * imgWidth_des_less * sizeof(uchar));

	dim3 block(8, 8);
	dim3 grid((imgWidth_des_less + block.x - 1) / block.x, (imgHeight_des_less + block.y - 1) / block.y);

	weightAddKerke_filter_N << <grid, block >> > (pDstImgData1, imgHeight_des_less, imgWidth_des_less, y_rato_less, x_rato_less,filter_cute,divide);
	hipDeviceSynchronize();

	//��GPU����������ݵ�CPU
	t = hipMemcpy(dstImg1.data, pDstImgData1, imgWidth_des_less * imgHeight_des_less * sizeof(uchar)*channels, hipMemcpyDeviceToHost);
	hipFree(cuArray_neareast_point);
	hipFree(pDstImgData1);
	//namedwindow("cuda_point�˲���", window_normal);
	imshow("cuda_point�˲���", dstImg1);
	waitKey(0);
	return dstImg1.clone();
}

int main()
{   //δ��ԭͼ�����˲�����
	Mat input= imread("C:/Users/Administrator/Desktop/lena.jpg");
	Mat mat_sx=RCDeletion("", input, 0.5, 0.5,0);//��С=Ƿ����
	mat_sx =RCDeletion("", mat_sx, 2, 2,1);//����С���ͼ��ԭ��ȥ
	cvtColor(input,input, COLOR_BGR2GRAY);

	imshow("ԭͼ��", input);//ԭͼ
	imshow("���˲���ԭͼ��", mat_sx);

    //��ԭͼ�����˲�����
	Mat input1=filter_N("C:/Users/Administrator/Desktop/lena.jpg", input, 1);
	Mat mat_sx1 = RCDeletion("", input1, 0.5, 0.5,1);//��С=Ƿ����
	mat_sx1 = RCDeletion("", mat_sx1, 2, 2,1);//����С���ͼ��ԭ��ȥ
	imshow("���˲���ԭͼ��", mat_sx1);
	waitKey(0);
	return 0;
}
