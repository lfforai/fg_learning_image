#include "hip/hip_runtime.h"
#pragma once
#include "morphology.cuh"

texture <uchar, hipTextureType2D, hipReadModeElementType> refTex_corrode;//���ڼ���˫���Բ�ֵ	

hipArray* cuArray_corrode;//����CUDA����

//ͨ����
hipChannelFormatDesc cuDesc_corrode = hipCreateChannelDesc<uchar>();


//ͼ��ʴchange center
__device__ uchar change_center(int x, int y, Point_gpu* point_gpu,uchar* data,int len) {
	int x_N;
	int y_N;
	uchar result = 255;
	for (int i = 0; i < len; i++)
	{
		x_N = (int)(point_gpu[i].x + x);
		y_N = (int)(point_gpu[i].y + y);
		if (tex2D(refTex_corrode, x_N, y_N) < 255 && data[i]==255)//ÿ��point_gpuλ���ϵ����ض���Ҫ��255������õ㽫����ʴ��
		{
			result = 0;
			break;
		}
	}
	return  result;
}

//ͼ��ʴ
__global__ void corrodeKerkel(int* pDstImgData, int imgHeight_des_d, int imgWidth_des_d,Point_gpu* point_gpu,uchar* data,int len)
	{   //printf("threadIdx,x=%d",threadIdx.x);
		const int tidx = blockDim.x * blockIdx.x + threadIdx.x;
		const int tidy = blockDim.y * blockIdx.y + threadIdx.y;
		if (tidx < imgWidth_des_d && tidy < imgHeight_des_d)
		{
			int idx = tidy * imgWidth_des_d + tidx;
			pDstImgData[idx] = (int)change_center(tidx,tidy,point_gpu,data,len);
			//printf("value=%u,%d,%d,%f,%f \n", pDstImgData[idx], x1, y2, x_des, y_des);
		}
	}


//ͼ������change center
__device__ void change_expand(int* pDstImgData,int x, int y, int imgWidth_des_d, int imgHeight_des_d,Point_gpu* point_gpu,uchar* data, int len) {
	int x_N;
	int y_N;
	int idx;
	for (int i = 0; i < len; i++)
	{   
		x_N = (int)(point_gpu[i].x + x);
		y_N = (int)(point_gpu[i].y + y);
		if (-1 < x_N && x_N < imgWidth_des_d && -1 < y_N && y_N < imgHeight_des_d)
		{   idx = (int)(y_N * imgWidth_des_d + x_N);
		
			 // printf("%u,%u \n", pDstImgData[idx], data[i]);
			if (pDstImgData[idx]==0 && tex2D(refTex_corrode, x_N, y_N)==255 && data[i] == 0)
			{
				int a =255;
				atomicExch(pDstImgData+idx,a);
				//printf("a: \n");
			   /* pDstImgData[idx] = data[i];*/
			}
			else {
				int a = (int)data[i];
				atomicExch(pDstImgData + idx, a);
				//pDstImgData[idx] =(int) data[i];
			}
		}
	}
}

//ͼ������
__global__ void expandKerkel(int* pDstImgData, int imgHeight_des_d, int imgWidth_des_d, Point_gpu* point_gpu,uchar* data,int len)
{   //printf("threadIdx,x=%d",threadIdx.x);
	const int tidx = blockDim.x * blockIdx.x + threadIdx.x;
	const int tidy = blockDim.y * blockIdx.y + threadIdx.y;
	if (tidx < imgWidth_des_d && tidy < imgHeight_des_d)
	{
		int idx = tidy * imgWidth_des_d + tidx;
		if(tex2D(refTex_corrode,tidx,tidy)==255)//�����255�����ؼ���Ƿ���Ҫ����
			change_expand(pDstImgData,tidx, tidy, imgWidth_des_d,imgHeight_des_d,point_gpu,data,len);//��Ҫ����
		//printf("value=%u,%d,%d,%f,%f \n", pDstImgData[idx], x1, y2, x_des, y_des);
	}
}

//mode=0 ��ʴ��1=����
Mat morphology_gpu(char * path,int len,Point_gpu*  point_offset_N,uchar* data ,int mode) {
	se_tpye * se_obj = (se_tpye*)malloc(sizeof(se_tpye));
	se_obj->init(len, point_offset_N,data);

	Mat Lena = imread(path);
	cvtColor(Lena, Lena, COLOR_BGR2GRAY);//ת��Ϊ�Ҷ�ͼ
	threshold(Lena, Lena, 100, 255, 0);
	image_show(Lena,1,"ԭͼ");
	
    int x_rato_less = 1.0;
	int y_rato_less = 1.0;

	int imgWidth_src = Lena.cols;//ԭͼ���
	int imgHeight_src = Lena.rows;//ԭͼ���
	int channels = Lena.channels();


	int imgWidth_des_less = floor(imgWidth_src * x_rato_less);//��Сͼ��
	int imgHeight_des_less = floor(imgHeight_src * y_rato_less);//��Сͼ��

	//����1��������
	hipError_t t;
	refTex_corrode.addressMode[0] = hipAddressModeBorder;
	refTex_corrode.addressMode[1] = hipAddressModeBorder;
	refTex_corrode.normalized = false;
	refTex_corrode.filterMode = hipFilterModePoint;
	//��cuArray������
	hipMallocArray(&cuArray_corrode, &cuDesc_corrode, imgWidth_src, imgHeight_src);
	t = hipBindTextureToArray(refTex_corrode, cuArray_corrode);
	//�������ݵ�hipArray
	t = hipMemcpyToArray(cuArray_corrode, 0, 0, Lena.data, imgWidth_src * imgHeight_src * sizeof(uchar), hipMemcpyHostToDevice);

	//��������Ժ���cpu��ͼ��
	Mat dstImg1 = Mat::zeros(imgHeight_des_less, imgWidth_des_less, CV_32SC1);//��С

	//��������Ժ���cuda�ϵ�ͼ��
	int* pDstImgData1 = NULL;
	t = hipMalloc(&pDstImgData1, imgHeight_des_less * imgWidth_des_less * sizeof(int));

	dim3 block(16, 16);
	dim3 grid((imgWidth_des_less + block.x - 1) / block.x, (imgHeight_des_less + block.y - 1) / block.y);

	if (mode==0)
	   corrodeKerkel << <grid, block >> > (pDstImgData1,imgHeight_des_less,imgWidth_des_less,se_obj->point_offset,se_obj->data,len);
	if (mode==1)
	   expandKerkel << <grid, block >> > (pDstImgData1, imgHeight_des_less, imgWidth_des_less, se_obj->point_offset,se_obj->data,len);
	hipDeviceSynchronize();

	//��GPU����������ݵ�CPU
	t = hipMemcpy(dstImg1.data, pDstImgData1, imgWidth_des_less * imgHeight_des_less * sizeof(int)*channels, hipMemcpyDeviceToHost);
	hipFree(cuArray_corrode);
	hipFree(pDstImgData1);
	stringstream ss;
	ss << len;
	string mark;
	ss >> mark;
	string ret = string("��ʴ�Ժ��ͼ") + mark;
	image_show(dstImg1,1,ret.c_str());
	//namedWindow("cuda_point�����ֵ��", WINDOW_NORMAL);
	//imshow("��ʴ�Ժ��ͼ��", dstImg1);
	//imwrite("C:/Users/Administrator/Desktop/ͼƬ/Gray_Image0.jpg", dstImg1);
	return dstImg1.clone();
}

//M�ǳ���N�ǿ�
Point_gpu* set_Point_gpu(int M, int N) {
	Point_gpu*  point_offset_N = (Point_gpu*)malloc(sizeof(Point_gpu) * M*N);
	int M_center = (int)M / 2;
	int N_center = (int)N / 2;
	for (size_t i = 0; i < M; i++)
	{
		for (size_t j = 0; j < N; j++)
		{
			point_offset_N[i*N + j].x = (int)j - N_center;
			point_offset_N[i*N + j].y = (int)i - M_center;
			//cout <<i<<"|"<<j<< endl;
			//cout<<point_offset_N[i*N + j].x <<endl;
			//cout<< point_offset_N[i*N + j].y << endl;
			//cout<<"--------------------------"<<endl;
		}
	}
	return point_offset_N;
}

//��ʴ��
uchar* set_Point_data(int M, int N) {
	uchar*  data = (uchar*)malloc(sizeof(uchar) * M*N);
	int M_center = (int)M / 2;
	int N_center = (int)N / 2;
	for (size_t i = 0; i < M; i++)
	{
		for (size_t j = 0; j < N; j++)
		{
			data[i*N + j]= 255;
		}
	}
	return data;
}

//������
uchar* set_Point_data_pz(int M, int N) {
	uchar*  data = (uchar*)malloc(sizeof(uchar)* M*N);
    data[0] = 0;
	data[1] = 255;
	data[2] = 0;
	data[3] = 255;
	data[4] = 255;
	data[5] = 255;
	data[6] = 0;
	data[7] = 255;
	data[8] = 0;
	return data;
}

void morphology_test(int M, int N,int mode)
{
	if(mode==0)
	{ Point_gpu* point_offset_N=set_Point_gpu(M,N);
	  uchar* data=set_Point_data(M,N);
	  morphology_gpu("C:/Users/Administrator/Desktop/opencv/m486.png", M*N, point_offset_N,data,0);
	}
	
	if(mode == 1)
	{ Point_gpu * point_offset_N =set_Point_gpu(M, N);;
	  //uchar* data = set_Point_data_pz(M,N);
	  uchar* data = set_Point_data(M, N);
	  morphology_gpu("C:/Users/Administrator/Desktop/opencv/font.png", M*N, point_offset_N, data, 1);
	}

}

//2����ֵͼ����߼���������,Ĭ�϶�ֵ����255��0����
Mat AND_two(const Mat& A, const Mat& B,uchar min,uchar max) {//������
	Mat A_N =A.clone();
	A_N.convertTo(A_N, CV_8U);
	Mat B_N =B.clone();
	B_N.convertTo(B_N, CV_8U);
	Mat result = Mat::zeros(A_N.size(), CV_8U);

	int N = A.cols;
	int M = A.rows;
	for (size_t i = 0; i <M; i++)
		{  for (size_t j = 0; j <N; j++)
			   { 
				 if (A_N.at<uchar>(i, j) == max && B_N.at<uchar>(i, j) == max)
				 {
					 result.at<uchar>(i, j) = max;

				 }
				 else {
					 result.at<uchar>(i, j) = min;
				 }
			   }
		  }

	return result.clone();
}

Mat OR_two(const Mat& A, const Mat& B, uchar min, uchar max) {//������
	Mat A_N = A.clone();
	A_N.convertTo(A_N, CV_8U);
	Mat B_N = B.clone();
	B_N.convertTo(B_N, CV_8U);
	Mat result= Mat::zeros(A_N.size(), CV_8U);

	int N = A.cols;
	int M = A.rows;
	for (size_t i = 0; i < M; i++)
	{
		for (size_t j = 0; j < N; j++)
		{
			if (A_N.at<uchar>(i, j) == min && B_N.at<uchar>(i, j) == min)
			{
				result.at<uchar>(i, j) = min;

			}
			else {
				result.at<uchar>(i, j) = max;
			}
		}
	}

	return result.clone();
}

Mat NOT_two(const Mat& A, uchar min, uchar max) {//������
	Mat A_N = A.clone();
	A_N.convertTo(A_N, CV_8U);

	Mat result=Mat::zeros(A_N.size(),CV_8U);

	int N = A.cols;
	int M = A.rows;
	for (size_t i = 0; i < M; i++)
	{
		for (size_t j = 0; j < N; j++)
		{
			if (A_N.at<uchar>(i, j)== min)
			{
				result.at<uchar>(i, j) = max;

			}
			else {
				result.at<uchar>(i, j) = min;
			}
		}
	}

	return result.clone();
}

Mat AND_NOT_two(const Mat& A, const Mat& B, uchar min, uchar max) {//B�����㣬����A��
	Mat mide = NOT_two(B,0,255);
	image_show(mide, 1, "mide");
	Mat result = AND_two(A,mide,0,255);
	return result.clone();
}

Mat XOR_two(const Mat& A, const Mat& B, uchar min , uchar max) {//������, ���ڣ�A��B�������޳�A��B
	Mat A_N = A.clone();
	A_N.convertTo(A_N, CV_8U);
	Mat B_N = B.clone();
	B_N.convertTo(B_N, CV_8U);
	Mat result;
	result.convertTo(result, CV_8U);

	int N = A.cols;
	int M = A.rows;
	for (size_t i = 0; i < M; i++)
	{
		for (size_t j = 0; j < N; j++)
		{
			if (A_N.at<uchar>(i, j)!=B_N.at<uchar>(i, j))
			{
				result.at<uchar>(i, j) = max;

			}
			else {
				result.at<uchar>(i, j) = min;
			}
		}
	}
	return result.clone();
}

//��9.5
void man_test() {
	Mat Lena = imread("C:/Users/Administrator/Desktop/opencv/man.png");
	cvtColor(Lena, Lena, COLOR_BGR2GRAY);//ת��Ϊ�Ҷ�ͼ
	threshold(Lena, Lena, 100, 255, 0);
	//image_show(Lena,1, "ԭͼ");
	
	int M = 5;
	int N = 5;
	Point_gpu* point=set_Point_gpu(M,N);
	uchar* data=set_Point_data(M, N);
	Mat mide=morphology_gpu("C:/Users/Administrator/Desktop/opencv/man.png", M*N,point, data, 0);
   
	Mat result=AND_NOT_two(Lena, mide,0,255);
	result.convertTo(result, CV_32F);
	image_show(result,1,"���");
}

void chapter9() {
	man_test();
}