#include "hip/hip_runtime.h"
#pragma once
#include "morphology.cuh"

texture <uchar, hipTextureType2D, hipReadModeElementType> refTex_corrode;//���ڼ���˫���Բ�ֵ	

hipArray* cuArray_corrode;//����CUDA����

//ͨ����
hipChannelFormatDesc cuDesc_corrode = hipCreateChannelDesc<uchar>();


//ͼ��ʴchange center
__device__ uchar change_center(int x, int y, Point_gpu* point_gpu, int len) {
	int x_N;
	int y_N;
	uchar result = 255;
	for (int i = 0; i < len; i++)
	{
		x_N = (int)(point_gpu[i].x + x);
		y_N = (int)(point_gpu[i].y + y);
		if (tex2D(refTex_corrode, x_N, y_N) < 255)//ÿ��point_gpuλ���ϵ����ض���Ҫ��255������õ㽫����ʴ��
		{
			result = 0;
			break;
		}
	}
	return  result;
}

//ͼ��ʴ
__global__ void corrodeKerkel(int* pDstImgData, int imgHeight_des_d, int imgWidth_des_d,Point_gpu* point_gpu,int len)
	{   //printf("threadIdx,x=%d",threadIdx.x);
		const int tidx = blockDim.x * blockIdx.x + threadIdx.x;
		const int tidy = blockDim.y * blockIdx.y + threadIdx.y;
		if (tidx < imgWidth_des_d && tidy < imgHeight_des_d)
		{
			int idx = tidy * imgWidth_des_d + tidx;
			pDstImgData[idx] = (int)change_center(tidx,tidy,point_gpu,len);
			//printf("value=%u,%d,%d,%f,%f \n", pDstImgData[idx], x1, y2, x_des, y_des);
		}
	}


//ͼ������change center
__device__ void change_expand(int* pDstImgData,int x, int y, int imgWidth_des_d, int imgHeight_des_d,Point_gpu* point_gpu,uchar* data, int len) {
	int x_N;
	int y_N;
	int idx;
	for (int i = 0; i < len; i++)
	{   
		x_N = (int)(point_gpu[i].x + x);
		y_N = (int)(point_gpu[i].y + y);
		if (-1 < x_N && x_N < imgWidth_des_d && -1 < y_N && y_N < imgHeight_des_d)
		{   idx = (int)(y_N * imgWidth_des_d + x_N);
		
			 // printf("%u,%u \n", pDstImgData[idx], data[i]);
			if (tex2D(refTex_corrode, x_N, y_N)==255 && data[i] == 0)
			{
				int a =255;
				atomicExch(pDstImgData+idx,a);
				//printf("a: \n");
			   /* pDstImgData[idx] = data[i];*/
			}
			else {
				int a = (int)data[i];
				atomicExch(pDstImgData + idx, a);
				//pDstImgData[idx] =(int) data[i];
			}
		}
	}
}

//ͼ������
__global__ void expandKerkel(int* pDstImgData, int imgHeight_des_d, int imgWidth_des_d, Point_gpu* point_gpu,uchar* data,int len)
{   //printf("threadIdx,x=%d",threadIdx.x);
	const int tidx = blockDim.x * blockIdx.x + threadIdx.x;
	const int tidy = blockDim.y * blockIdx.y + threadIdx.y;
	if (tidx < imgWidth_des_d && tidy < imgHeight_des_d)
	{
		int idx = tidy * imgWidth_des_d + tidx;
		if(tex2D(refTex_corrode,tidx,tidy)==255)//�����255�����ؼ���Ƿ���Ҫ����
			change_expand(pDstImgData,tidx, tidy, imgWidth_des_d,imgHeight_des_d,point_gpu,data,len);//��Ҫ����
		//printf("value=%u,%d,%d,%f,%f \n", pDstImgData[idx], x1, y2, x_des, y_des);
	}
}

//mode=0 ��ʴ��1=����
void morphology_gpu(char * path,int len,Point_gpu*  point_offset_N,uchar* data ,int mode) {
	se_tpye * se_obj = (se_tpye*)malloc(sizeof(se_tpye));
	se_obj->init(len, point_offset_N,data);

	Mat Lena = imread(path);
	cvtColor(Lena, Lena, COLOR_BGR2GRAY);//ת��Ϊ�Ҷ�ͼ
	threshold(Lena, Lena, 100, 255, 0);
	image_show(Lena,0.5,"ԭͼ");
	
    int x_rato_less = 1.0;
	int y_rato_less = 1.0;

	int imgWidth_src = Lena.cols;//ԭͼ���
	int imgHeight_src = Lena.rows;//ԭͼ���
	int channels = Lena.channels();


	int imgWidth_des_less = floor(imgWidth_src * x_rato_less);//��Сͼ��
	int imgHeight_des_less = floor(imgHeight_src * y_rato_less);//��Сͼ��

	//����1��������
	hipError_t t;
	refTex_corrode.addressMode[0] = hipAddressModeBorder;
	refTex_corrode.addressMode[1] = hipAddressModeBorder;
	refTex_corrode.normalized = false;
	refTex_corrode.filterMode = hipFilterModePoint;
	//��cuArray������
	hipMallocArray(&cuArray_corrode, &cuDesc_corrode, imgWidth_src, imgHeight_src);
	t = hipBindTextureToArray(refTex_corrode, cuArray_corrode);
	//�������ݵ�hipArray
	t = hipMemcpyToArray(cuArray_corrode, 0, 0, Lena.data, imgWidth_src * imgHeight_src * sizeof(uchar), hipMemcpyHostToDevice);

	//��������Ժ���cpu��ͼ��
	Mat dstImg1 = Mat::zeros(imgHeight_des_less, imgWidth_des_less, CV_32SC1);//��С

	//��������Ժ���cuda�ϵ�ͼ��
	int* pDstImgData1 = NULL;
	t = hipMalloc(&pDstImgData1, imgHeight_des_less * imgWidth_des_less * sizeof(int));

	dim3 block(16, 16);
	dim3 grid((imgWidth_des_less + block.x - 1) / block.x, (imgHeight_des_less + block.y - 1) / block.y);

	if (mode==0)
	   corrodeKerkel << <grid, block >> > (pDstImgData1,imgHeight_des_less,imgWidth_des_less,se_obj->point_offset,len);
	if (mode==1)
	   expandKerkel << <grid, block >> > (pDstImgData1, imgHeight_des_less, imgWidth_des_less, se_obj->point_offset, se_obj->data,len);
	hipDeviceSynchronize();

	//��GPU����������ݵ�CPU
	t = hipMemcpy(dstImg1.data, pDstImgData1, imgWidth_des_less * imgHeight_des_less * sizeof(int)*channels, hipMemcpyDeviceToHost);
	hipFree(cuArray_corrode);
	hipFree(pDstImgData1);
	stringstream ss;
	ss << len;
	string mark;
	ss >> mark;
	string ret = string("��ʴ�Ժ��ͼ") + mark;
	image_show(dstImg1,0.5,ret.c_str());
	//namedWindow("cuda_point�����ֵ��", WINDOW_NORMAL);
	//imshow("��ʴ�Ժ��ͼ��", dstImg1);
	//imwrite("C:/Users/Administrator/Desktop/ͼƬ/Gray_Image0.jpg", dstImg1);
}

//M�ǳ���N�ǿ�
Point_gpu* set_Point_gpu(int M, int N) {
	Point_gpu*  point_offset_N = (Point_gpu*)malloc(sizeof(Point_gpu) * M*N);
	int M_center = (int)M / 2;
	int N_center = (int)N / 2;
	for (size_t i = 0; i < M; i++)
	{
		for (size_t j = 0; j < N; j++)
		{
			point_offset_N[i*N + j].x = (int)j - N_center;
			point_offset_N[i*N + j].y = (int)i - M_center;
			//cout <<i<<"|"<<j<< endl;
			//cout<<point_offset_N[i*N + j].x <<endl;
			//cout<< point_offset_N[i*N + j].y << endl;
			//cout<<"--------------------------"<<endl;
		}
	}
	return point_offset_N;
}

//��ʴ��
uchar* set_Point_data(int M, int N) {
	uchar*  data = (uchar*)malloc(sizeof(uchar) * M*N);
	int M_center = (int)M / 2;
	int N_center = (int)N / 2;
	for (size_t i = 0; i < M; i++)
	{
		for (size_t j = 0; j < N; j++)
		{
			data[i*N + j]= 255;
		}
	}
	return data;
}

//������
uchar* set_Point_data_pz(int M, int N) {
	uchar*  data = (uchar*)malloc(sizeof(uchar)* M*N);
    data[0] = 0;
	data[1] = 255;
	data[2] = 0;
	data[3] = 255;
	data[4] = 255;
	data[5] = 255;
	data[6] = 0;
	data[7] = 255;
	data[8] = 0;
	return data;
}

void morphology_test(int M, int N,int mode)
{
	if(mode==0)
	{ Point_gpu* point_offset_N=set_Point_gpu(M,N);
	  uchar* data=set_Point_data(M,N);
	  morphology_gpu("C:/Users/Administrator/Desktop/opencv/m486.png", M*N, point_offset_N,data,0);
	}
	
	if(mode == 1)
	{ Point_gpu * point_offset_N =set_Point_gpu(M, N);;
	  //uchar* data = set_Point_data_pz(M,N);
	  uchar* data = set_Point_data(M, N);
	  morphology_gpu("C:/Users/Administrator/Desktop/opencv/font.png", M*N, point_offset_N, data, 1);
	}
}