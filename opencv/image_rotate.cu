#include "hip/hip_runtime.h"
#pragma once
#include "image_rotate.cuh"

//һ����ת�任
// Texture reference for 2D float texture
texture <uchar, hipTextureType2D, hipReadModeElementType> refTex;

//����CUDA����
hipArray* cuArray;//���ڼ������point��ֵ

//ͨ����
hipChannelFormatDesc cuDesc = hipCreateChannelDesc<uchar>();

////////////////////////////////////////////////////////////////////////////////
//! Transform an image using texture lookups
//! @param outputData  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void transformKernel(uchar *outputData,
	int height, 
	int width,
	const float theta)
{
	// calculate normalized texture coordinates
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	float u = (float)x - (float)width  / 2.0;
	float v = (float)y - (float)height / 2.0;
	float tu = u * cosf(theta) - v * sinf(theta);
	float tv = v * cosf(theta) + u * sinf(theta);

	tu /= (float)width;
	tv /= (float)height;

	// read from texture and write to global memory
	outputData[y*width + x] = (uchar)(tex2D(refTex, tu + 0.5f, tv + 0.5f));
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
Mat image_rotate_point(char* path,Mat lena_o,int ifhd)
{
	const float angle = -0.5f;        // angle to rotate image by (in radians)
	Mat Lena;
	if (strlen(path) == 0) {
		Lena = lena_o.clone();
	}
	else {
		Lena = imread(path);
	}

	if (ifhd == 0)//���ǻҶ�ͼҪ����ת��
		cvtColor(Lena, Lena, COLOR_BGR2GRAY);//ת��Ϊ�Ҷ�ͼ

	float x_rato_less = 1.0;
	float y_rato_less = 1.0;

	int imgWidth_src = Lena.cols;//ԭͼ���
	int imgHeight_src = Lena.rows;//ԭͼ���
	int channels = Lena.channels();

	int imgWidth_des_less = floor(imgWidth_src * x_rato_less);//��Сͼ��
	int imgHeight_des_less = floor(imgHeight_src * y_rato_less);//��Сͼ��

   //����1��������
	hipError_t t;
	refTex.addressMode[0] = hipAddressModeBorder;
	refTex.addressMode[1] = hipAddressModeBorder;
	refTex.normalized = true;
	refTex.filterMode = hipFilterModePoint;
	//��cuArray������
	hipMallocArray(&cuArray, &cuDesc, imgWidth_src, imgHeight_src);
	t = hipBindTextureToArray(refTex, cuArray);
	//�������ݵ�hipArray
	t = hipMemcpyToArray(cuArray, 0, 0, Lena.data, imgWidth_src * imgHeight_src * sizeof(uchar), hipMemcpyHostToDevice);

	//��������Ժ���cpu��ͼ��
	Mat dstImg1 = Mat::zeros(imgHeight_des_less, imgWidth_des_less, CV_8UC1);//��С

	//��������Ժ���cuda�ϵ�ͼ��
	uchar* pDstImgData1 = NULL;
	t = hipMalloc(&pDstImgData1, imgHeight_des_less * imgWidth_des_less * sizeof(uchar));

	dim3 block(8, 8);
	dim3 grid((imgWidth_des_less + block.x - 1) / block.x, (imgHeight_des_less + block.y - 1) / block.y);

	transformKernel << <grid, block >> > (pDstImgData1, imgHeight_des_less, imgWidth_des_less, angle);
	hipDeviceSynchronize();

	//��GPU����������ݵ�CPU
	t = hipMemcpy(dstImg1.data, pDstImgData1, imgWidth_des_less * imgHeight_des_less * sizeof(uchar)*channels, hipMemcpyDeviceToHost);
	hipFree(cuArray);
	hipFree(pDstImgData1);
	//imshow("ԭͼ��", Lena);
	//imshow("��ת�Ժ��ͼ��", dstImg1);
	return dstImg1.clone();
	//imwrite("C:/Users/Administrator/Desktop/ͼƬ/Gray_Image0.jpg", dstImg1);
	/*waitKey(0);*/
}

//����������ƶ�
// Texture reference for 2D float texture
texture <uchar, hipTextureType2D, hipReadModeElementType> refTex_move;

//����CUDA����
hipArray* cuArray_move;//���ڼ������point��ֵ

__global__ void transformKernel_move(uchar *outputData,
	int height,
	int width,
	int x_move,
	int y_move)
{
	// calculate normalized texture coordinates
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	int tu = x - x_move;
	int tv = y - y_move;

	// read from texture and write to global memory
	outputData[y*width + x] = tex2D(refTex_move,tu,tv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
Mat image_move_point(char* path, Mat lena_o, int ifhd, int x_move,int y_move)
{     
	Mat Lena;
	if (strlen(path) == 0) {
		Lena = lena_o.clone();
	}
	else {
		Lena = imread(path);
	}

	if (ifhd == 0)//���ǻҶ�ͼҪ����ת��
		cvtColor(Lena, Lena, COLOR_BGR2GRAY);//ת��Ϊ�Ҷ�ͼ

	float x_rato_less = 1.0;
	float y_rato_less = 1.0;

	int imgWidth_src = Lena.cols;//ԭͼ���
	int imgHeight_src = Lena.rows;//ԭͼ���
	int channels = Lena.channels();

	int imgWidth_des_less = floor(imgWidth_src * x_rato_less);//��Сͼ��
	int imgHeight_des_less = floor(imgHeight_src * y_rato_less);//��Сͼ��

   //����1��������
	hipError_t t;
	refTex.addressMode[0] = hipAddressModeBorder;
	refTex.addressMode[1] = hipAddressModeBorder;
	refTex.normalized = false;
	refTex.filterMode = hipFilterModePoint;
	//��cuArray������
	hipMallocArray(&cuArray_move, &cuDesc, imgWidth_src, imgHeight_src);
	t = hipBindTextureToArray(refTex_move, cuArray_move);
	//�������ݵ�hipArray
	t = hipMemcpyToArray(cuArray_move, 0, 0, Lena.data, imgWidth_src * imgHeight_src * sizeof(uchar), hipMemcpyHostToDevice);

	//��������Ժ���cpu��ͼ��
	Mat dstImg1 = Mat::zeros(imgHeight_des_less, imgWidth_des_less, CV_8UC1);//��С

	//��������Ժ���cuda�ϵ�ͼ��
	uchar* pDstImgData1 = NULL;
	t = hipMalloc(&pDstImgData1, imgHeight_des_less * imgWidth_des_less * sizeof(uchar));

	dim3 block(8, 8);
	dim3 grid((imgWidth_des_less + block.x - 1) / block.x, (imgHeight_des_less + block.y - 1) / block.y);

	transformKernel_move << <grid, block >> > (pDstImgData1, imgHeight_des_less, imgWidth_des_less,x_move,y_move);
	hipDeviceSynchronize();

	//��GPU����������ݵ�CPU
	t = hipMemcpy(dstImg1.data, pDstImgData1, imgWidth_des_less * imgHeight_des_less * sizeof(uchar)*channels, hipMemcpyDeviceToHost);
	hipFree(cuArray_move);
	hipFree(pDstImgData1);
	//imshow("ԭͼ��", Lena);
	//imshow("�ƶ��Ժ��ͼ��", dstImg1);
	return dstImg1.clone();
	//imwrite("C:/Users/Administrator/Desktop/ͼƬ/Gray_Image0.jpg", dstImg1);
	/*waitKey(0);*/
}

Mat image_rotate_point_GPU(char* path, Mat lena_o, int ifhd) {
	return image_rotate_point(path, lena_o, ifhd);
}

Mat image_move_point_GPU(char* path, Mat lena_o, int ifhd, int x_move,int y_move) {
	return image_move_point(path, lena_o,ifhd,x_move,y_move);
}