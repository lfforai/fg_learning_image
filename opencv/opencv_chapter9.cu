#include "hip/hip_runtime.h"
#pragma once
#include "opencv_chapter9.cuh"
texture <uchar, hipTextureType2D, hipReadModeElementType> refTex_corrode;//���ڼ���˫���Բ�ֵ

hipArray* cuArray_corrode;//����CUDA����

//ͨ����
hipChannelFormatDesc cuDesc_corrode = hipCreateChannelDesc<uchar>();

//ͼ��ʴchange center
__device__ uchar change_center(int x, int y, Point_gpu* point_gpu,uchar* data,int len) {
	
		int x_N;
		int y_N;
		uchar result = 255;
		for (int i = 0; i < len; i++)
		{
			x_N = (int)(point_gpu[i].x + x);
			y_N = (int)(point_gpu[i].y + y);
			if (tex2D(refTex_corrode, x_N, y_N) < 255 && data[i] == 255)//ÿ��point_gpuλ���ϵ����ض���Ҫ��255������õ㽫����ʴ��
			{   //����õ�data��i��=0,��ʾ���Ǹ�ʴ��������
				result = 0;
				break;
			}
		}
		return  result;	
}

__device__ uchar change_center_catch(int x, int y, Point_gpu* point_gpu, uchar* data, int len) {
		int x_N;
		int y_N;
		uchar result;
		for (int i = 0; i < len; i++)
		{
			x_N = (int)(point_gpu[i].x + x);
			y_N = (int)(point_gpu[i].y + y);
			if (tex2D(refTex_corrode, x_N, y_N) == 255 && data[i] == 255)//ÿ��point_gpuλ���ϵ����ض���Ҫ��255��
			{   //����õ�data��i��=255,��ʾ�߱��˿�����������
				result = 255;
			}

			if (tex2D(refTex_corrode, x_N, y_N) == 0 && data[i] == 255)//���������ƥ����������0�㻹ά��ԭ0
			{   //����õ�������data��i����һ��,��ʾ���Ǹ�ʴ������������0����Ȼ�����	
		    	result = 0;
				break;
			}
		}
		return  result;
}

//ͼ��ʴ
__global__ void corrodeKerkel(int* pDstImgData, int imgHeight_des_d, int imgWidth_des_d,Point_gpu* point_gpu,uchar* data,int len)
	{   //printf("threadIdx,x=%d",threadIdx.x);
		const int tidx = blockDim.x * blockIdx.x + threadIdx.x;
		const int tidy = blockDim.y * blockIdx.y + threadIdx.y;
		if (tidx < imgWidth_des_d && tidy < imgHeight_des_d)
		{
			int idx = tidy * imgWidth_des_d + tidx;
			int index;
			
			if (data[(int)(len / 2)] == 255)//Ѱ��255���и�ʴ����
			{
				//printf("%u \n:", data[(int)(len / 2)]);
				if (tex2D(refTex_corrode, tidx, tidy) == 255)
					pDstImgData[idx] = (int)change_center(tidx, tidy, point_gpu, data, len);
			}
			else if (data[(int)(len / 2)] == 0) {
				if (tex2D(refTex_corrode, tidx, tidy) == 0)//Ѱ��0���л��л����е����
					pDstImgData[idx] = (int)change_center_catch(tidx, tidy, point_gpu, data, len);
			}
			//printf("value=%u,%d,%d,%f,%f \n", pDstImgData[idx], x1, y2, x_des, y_des);
		}
	}


//ͼ������change center
__device__ void change_expand(int* pDstImgData,int x, int y, int imgWidth_des_d, int imgHeight_des_d,Point_gpu* point_gpu,uchar* data, int len) {
	int x_N;
	int y_N;
	int idx;
	int a;
	for (int i = 0; i < len; i++)
	{   
		x_N = (int)(point_gpu[i].x + x);
		y_N = (int)(point_gpu[i].y + y);
		a = (int)tex2D(refTex_corrode, x_N, y_N);
		if (~(y_N == 0 && x_N ==0) && -1 < x_N && x_N < imgWidth_des_d && -1 < y_N && y_N < imgHeight_des_d && data[i]==255 && a==0)
		{   idx = (int)(y_N * imgWidth_des_d + x_N);
		    a = 255;
			atomicExch(pDstImgData + idx, a);
		}
	}
}

//ͼ������
__global__ void expandKerkel(int* pDstImgData, int imgHeight_des_d, int imgWidth_des_d, Point_gpu* point_gpu,uchar* data,int len)
{   //printf("threadIdx,x=%d",threadIdx.x);
	const int tidx = blockDim.x * blockIdx.x + threadIdx.x;
	const int tidy = blockDim.y * blockIdx.y + threadIdx.y;
	if (tidx < imgWidth_des_d && tidy < imgHeight_des_d)
	{
		int idx = tidy * imgWidth_des_d + tidx;
		if(tex2D(refTex_corrode,tidx,tidy)==255)//�����255�����ؼ���Ƿ���Ҫ����
			change_expand(pDstImgData,tidx, tidy, imgWidth_des_d,imgHeight_des_d,point_gpu,data,len);//��Ҫ����
		//printf("value=%u,%d,%d,%f,%f \n", pDstImgData[idx], x1, y2, x_des, y_des);
	}
}



//mode=0 ��ʴ��1=����
Mat morphology_gpu(char * path,int len,Point_gpu*  point_offset_N,uchar* data ,int mode) {
	se_tpye * se_obj = (se_tpye*)malloc(sizeof(se_tpye));
	se_obj->init(len, point_offset_N,data);

	Mat Lena = imread(path);
	cvtColor(Lena, Lena, COLOR_BGR2GRAY);//ת��Ϊ�Ҷ�ͼ
	threshold(Lena, Lena, 100, 255, 0);
	image_show(Lena,1,"ԭͼ");
	
	int x_rato_less = 1.0;
	int y_rato_less = 1.0;

	int imgWidth_src = Lena.cols;//ԭͼ���
	int imgHeight_src = Lena.rows;//ԭͼ���
	int channels = Lena.channels();

	int imgWidth_des_less = floor(imgWidth_src * x_rato_less);//��Сͼ��
	int imgHeight_des_less = floor(imgHeight_src * y_rato_less);//��Сͼ��

	//����1��������
	hipError_t t;
	refTex_corrode.addressMode[0] = hipAddressModeBorder;
	refTex_corrode.addressMode[1] = hipAddressModeBorder;
	refTex_corrode.normalized = false;
	refTex_corrode.filterMode = hipFilterModePoint;
	//��cuArray������
	hipMallocArray(&cuArray_corrode, &cuDesc_corrode, imgWidth_src, imgHeight_src);
	t = hipBindTextureToArray(refTex_corrode, cuArray_corrode);
	//�������ݵ�hipArray
	t = hipMemcpyToArray(cuArray_corrode, 0, 0, Lena.data, imgWidth_src * imgHeight_src * sizeof(uchar), hipMemcpyHostToDevice);

	//��������Ժ���cpu��ͼ��
	Lena.convertTo(Lena, CV_32S);
	Mat dstImg1 = Mat::zeros(imgHeight_des_less, imgWidth_des_less, CV_32SC1);//��С

	//��������Ժ���cuda�ϵ�ͼ��
	int* pDstImgData1 = NULL;
	t = hipMalloc(&pDstImgData1, imgHeight_des_less * imgWidth_des_less * sizeof(int));
	t = hipMemcpy(pDstImgData1, Lena.data, imgWidth_des_less * imgHeight_des_less * sizeof(int)*channels, hipMemcpyHostToDevice);

	dim3 block(16, 16);
	dim3 grid((imgWidth_des_less + block.x - 1) / block.x, (imgHeight_des_less + block.y - 1) / block.y);

	if (mode == 0)
		corrodeKerkel << <grid, block >> > (pDstImgData1, imgHeight_des_less, imgWidth_des_less, se_obj->point_offset, se_obj->data, len);
	if (mode == 1)
		expandKerkel << <grid, block >> > (pDstImgData1, imgHeight_des_less, imgWidth_des_less, se_obj->point_offset, se_obj->data, len);
	hipDeviceSynchronize();

	//��GPU����������ݵ�CPU
	t = hipMemcpy(dstImg1.data, pDstImgData1, imgWidth_des_less * imgHeight_des_less * sizeof(int)*channels, hipMemcpyDeviceToHost);
	hipFree(cuArray_corrode);
	hipFree(pDstImgData1);
	stringstream ss;
	ss << len;
	string mark;
	ss >> mark;
	string ret = string("��ʴ�Ժ��ͼ") + mark;
	image_show(dstImg1, 1, ret.c_str());
	//namedWindow("cuda_point�����ֵ��", WINDOW_NORMAL);
	//imshow("��ʴ�Ժ��ͼ��", dstImg1);
	//imwrite("C:/Users/Administrator/Desktop/ͼƬ/Gray_Image0.jpg", dstImg1);
	return dstImg1.clone();
}

//mode=0 ��ʴ��1=����
Mat morphology_gpu_Mat(const Mat& image, int len, Point_gpu*  point_offset_N, uchar* data, int mode) {
	se_tpye * se_obj = (se_tpye*)malloc(sizeof(se_tpye));
	se_obj->init(len, point_offset_N, data);

	Mat Lena = image.clone();
	Lena.convertTo(Lena, CV_8U);
	//image_show(Lena, 1, "ԭͼ");

	int x_rato_less = 1.0;
	int y_rato_less = 1.0;

	int imgWidth_src = Lena.cols;//ԭͼ���
	int imgHeight_src = Lena.rows;//ԭͼ���
	int channels = Lena.channels();

	int imgWidth_des_less = floor(imgWidth_src * x_rato_less);//��Сͼ��
	int imgHeight_des_less = floor(imgHeight_src * y_rato_less);//��Сͼ��

	//����1��������
	hipError_t t;
	refTex_corrode.addressMode[0] = hipAddressModeBorder;
	refTex_corrode.addressMode[1] = hipAddressModeBorder;
	refTex_corrode.normalized = false;
	refTex_corrode.filterMode = hipFilterModePoint;
	//��cuArray������
	hipMallocArray(&cuArray_corrode, &cuDesc_corrode, imgWidth_src, imgHeight_src);
	t = hipBindTextureToArray(refTex_corrode, cuArray_corrode);
	//�������ݵ�hipArray
	t = hipMemcpyToArray(cuArray_corrode, 0, 0, Lena.data, imgWidth_src * imgHeight_src * sizeof(uchar), hipMemcpyHostToDevice);

	//��������Ժ���cpu��ͼ��
	Lena.convertTo(Lena, CV_32S);
	Mat dstImg1 = Mat::zeros(imgHeight_des_less, imgWidth_des_less, CV_32SC1);//��С

	//��������Ժ���cuda�ϵ�ͼ��
	int* pDstImgData1 = NULL;
	t = hipMalloc(&pDstImgData1,imgHeight_des_less * imgWidth_des_less * sizeof(int)); 
	t = hipMemcpy(pDstImgData1,Lena.data,imgWidth_des_less * imgHeight_des_less * sizeof(int)*channels, hipMemcpyHostToDevice);

	dim3 block(16, 16);
	dim3 grid((imgWidth_des_less + block.x - 1) / block.x, (imgHeight_des_less + block.y - 1) / block.y);

	if (mode == 0)
		corrodeKerkel << <grid, block >> > (pDstImgData1, imgHeight_des_less, imgWidth_des_less, se_obj->point_offset, se_obj->data, len);
	if (mode == 1)
		expandKerkel << <grid, block >> > (pDstImgData1, imgHeight_des_less, imgWidth_des_less, se_obj->point_offset, se_obj->data, len);
	hipDeviceSynchronize();

	//��GPU����������ݵ�CPU
	t = hipMemcpy(dstImg1.data, pDstImgData1, imgWidth_des_less * imgHeight_des_less * sizeof(int)*channels, hipMemcpyDeviceToHost);
	hipFree(cuArray_corrode);
	hipFree(pDstImgData1);
	stringstream ss;
	ss << len;
	string mark;
	ss >> mark;
	string ret = string("��ʴ�Ժ��ͼ") + mark;
	image_show(dstImg1, 1, ret.c_str());
	//namedWindow("cuda_point�����ֵ��", WINDOW_NORMAL);
	//imshow("��ʴ�Ժ��ͼ��", dstImg1);
	//imwrite("C:/Users/Administrator/Desktop/ͼƬ/Gray_Image0.jpg", dstImg1);
	return dstImg1.clone();
}


//M�ǳ���N�ǿ�
Point_gpu* set_Point_gpu(int M, int N) {
	Point_gpu*  point_offset_N = (Point_gpu*)malloc(sizeof(Point_gpu) * M*N);
	int M_center = (int)M / 2;
	int N_center = (int)N / 2;
	for (size_t i = 0; i < M; i++)
	{
		for (size_t j = 0; j < N; j++)
		{
			point_offset_N[i*N + j].x = (int)j - N_center;
			point_offset_N[i*N + j].y = (int)i - M_center;
			//cout <<i<<"|"<<j<< endl;
			//cout<<point_offset_N[i*N + j].x <<endl;
			//cout<< point_offset_N[i*N + j].y << endl;
			//cout<<"--------------------------"<<endl;
		}
	}
	return point_offset_N;
}


//M�ǳ���N�ǿ�,����һ���ڽ�Բ��-ƽ̹�ṹԪ
int* set_Point_data_circle(int M) 
{
	//�����ڲ�����
	auto ifin = [](int x_rows, int y_cols, int P_mide, int  Q_mide, int D0_radius)->bool {
		bool r = false;//������D0��
		if ((float)sqrt(pow(x_rows - P_mide, 2.0) + pow(y_cols - Q_mide, 2.0)) <=(float) D0_radius)
		{
			/*cout<< (float)sqrt(pow(x_rows - P_mide, 2.0) + pow(y_cols - Q_mide, 2.0)) <<endl;*/
			r = true;
		}
		return r;
	};

	int*  data = (int*)malloc(sizeof(int) * M*M);
	int M_center = (int)M / 2;
	int N_center = (int)M / 2;
	cout<< M_center <<endl;
	for(size_t i = 0; i < M; i++)
	{   for(size_t j = 0; j < M; j++)
		  {   if(ifin(j, i, M_center, N_center, M_center))
			    {data[i*M + j] =-255;//���ܺ��Ե�
			    }else{
				 data[i*M + j] =-1;//���Ժ��Ե�
                }
		  }
	}
	return data;
}


//��ʴ��
uchar* set_Point_data(int M, int N) {
	uchar*  data = (uchar*)malloc(sizeof(uchar) * M*N);
	int M_center = (int)M / 2;
	int N_center = (int)N / 2;
	for (size_t i = 0; i < M; i++)
	{
		for (size_t j = 0; j < N; j++)
		{
			data[i*N + j]= 255;
		}
	}
	return data;
}

//���С�������
uchar* set_Point_data_jz(int M, int N) {
	uchar*  data = (uchar*)malloc(sizeof(uchar) * M*N);
	int M_center = (int)M / 2;
	int N_center = (int)N / 2;
	for (size_t i = 0; i < M; i++)
	{
		for (size_t j = 0; j < N; j++)
		{
			data[i*N + j] = 0;
		}
	}
	return data;
}

//�Ǳ�׼����ˣ�������
uchar* set_Point_data_pz(int M, int N) {
	uchar*  data = (uchar*)malloc(sizeof(uchar)* M*N);
    data[0] = 0;
	data[1] = 255;
	data[2] = 0;
	data[3] = 255;
	data[4] = 255;
	data[5] = 255;
	data[6] = 0;
	data[7] = 255;
	data[8] = 0;
	return data;
}

void test() {
	int M = 3;
	int N = 3;

	Mat X = Mat::zeros(Size(5, 5), CV_8U);
	X.at<uchar>(0, 1) = 255;
	X.at<uchar>(0, 0) = 255;
	X.at<uchar>(0, 2) = 255;
	X.at<uchar>(1, 1) = 255;
	//X.at<uchar>(2, 4) = 255;
	//X.at<uchar>(3, 3) = 255;
	cout << X << endl;
	Point_gpu* point = set_Point_gpu(M, N);
	uchar* data = set_Point_data_pz(M, N);
	Mat mide = morphology_gpu_Mat(X, M*N, point, data, 1);
	cout << mide << endl;
	cout << "-------------------------" << endl;
	for (size_t i = 0; i < 1; i++)
	{
		mide = morphology_gpu_Mat(mide, M*N, point, data, 1);
	}
	//mide.convertTo(mide, CV_8U);
	cout << mide << endl;
	cout << "-------------------------" << endl;
}

void morphology_test(int M, int N,int mode)
{
	if(mode==0)
	{ Point_gpu* point_offset_N=set_Point_gpu(M,N);
	  uchar* data=set_Point_data(M,N);
	  morphology_gpu("C:/Users/Administrator/Desktop/opencv/m486.png", M*N, point_offset_N,data,0);
	}
	
	if(mode == 1)
	{ Point_gpu * point_offset_N =set_Point_gpu(M, N);;
	  uchar* data = set_Point_data_pz(M,N);
	  //uchar* data = set_Point_data(M, N);
	  morphology_gpu("C:/Users/Administrator/Desktop/opencv/font1.png", M*N, point_offset_N, data, 1);
	}

}

//2����ֵͼ����߼���������,Ĭ�϶�ֵ����255��0����
Mat AND_two(const Mat& A, const Mat& B,uchar min,uchar max) {//������
	Mat A_N =A.clone();
	A_N.convertTo(A_N, CV_8U);
	Mat B_N =B.clone();
	B_N.convertTo(B_N, CV_8U);
	Mat result = Mat::zeros(A_N.size(), CV_8U);

	int N = A.cols;
	int M = A.rows;
	for (size_t i = 0; i <M; i++)
		{  for (size_t j = 0; j <N; j++)
			   { 
				 if (A_N.at<uchar>(i, j) == max && B_N.at<uchar>(i, j) == max)
				 {
					 result.at<uchar>(i, j) = max;

				 }
				 else {
					 result.at<uchar>(i, j) = min;
				 }
			   }
		  }

	return result.clone();
}

Mat OR_two(const Mat& A, const Mat& B, uchar min, uchar max) {//������
	Mat A_N = A.clone();
	A_N.convertTo(A_N, CV_8U);
	Mat B_N = B.clone();
	B_N.convertTo(B_N, CV_8U);
	Mat result= Mat::zeros(A_N.size(), CV_8U);

	int N = A.cols;
	int M = A.rows;
	for (size_t i = 0; i < M; i++)
	{
		for (size_t j = 0; j < N; j++)
		{
			if (A_N.at<uchar>(i, j) == min && B_N.at<uchar>(i, j) == min)
			{
				result.at<uchar>(i, j) = min;

			}
			else {
				result.at<uchar>(i, j) = max;
			}
		}
	}

	return result.clone();
}

Mat NOT_two(const Mat& A, uchar min, uchar max) {//������
	Mat A_N = A.clone();
	A_N.convertTo(A_N, CV_8U);

	Mat result=Mat::zeros(A_N.size(),CV_8U);

	int N = A.cols;
	int M = A.rows;
	for (size_t i = 0; i < M; i++)
	{
		for (size_t j = 0; j < N; j++)
		{
			if (A_N.at<uchar>(i, j)== min)
			{
				result.at<uchar>(i, j) = max;

			}
			else {
				result.at<uchar>(i, j) = min;
			}
		}
	}

	return result.clone();
}

Mat AND_NOT_two(const Mat& A, const Mat& B, uchar min, uchar max) {//B�����㣬����A��
	Mat mide = NOT_two(B,0,255);
	image_show(mide, 1, "mide");
	Mat result = AND_two(A,mide,0,255);
	return result.clone();
}

Mat XOR_two(const Mat& A, const Mat& B, uchar min , uchar max) {//������, ���ڣ�A��B�������޳�A��B
	Mat A_N = A.clone();
	A_N.convertTo(A_N, CV_8U);
	Mat B_N = B.clone();
	B_N.convertTo(B_N, CV_8U);
	Mat result;
	result.convertTo(result, CV_8U);

	int N = A.cols;
	int M = A.rows;
	for (size_t i = 0; i < M; i++)
	{
		for (size_t j = 0; j < N; j++)
		{
			if (A_N.at<uchar>(i, j)!=B_N.at<uchar>(i, j))
			{
				result.at<uchar>(i, j) = max;

			}
			else {
				result.at<uchar>(i, j) = min;
			}
		}
	}
	return result.clone();
}

//��9.5
void man_test() {
	Mat Lena = imread("C:/Users/Administrator/Desktop/opencv/man.png");
	cvtColor(Lena, Lena, COLOR_BGR2GRAY);//ת��Ϊ�Ҷ�ͼ
	threshold(Lena, Lena, 100, 255, 0);
	//image_show(Lena,1, "ԭͼ");
	
	int M = 5;
	int N = 5;
	Point_gpu* point=set_Point_gpu(M,N);
	uchar* data=set_Point_data(M,N);
	Mat mide=morphology_gpu("C:/Users/Administrator/Desktop/opencv/man.png", M*N,point, data, 0);
   
	Mat result=AND_NOT_two(Lena, mide,0,255);
	result.convertTo(result, CV_32F);
	image_show(result,1,"���");
}

//��9.6
void remove_test() {
	Mat Lena = imread("C:/Users/Administrator/Desktop/opencv/yuan.png");
	cvtColor(Lena, Lena, COLOR_BGR2GRAY);//ת��Ϊ�Ҷ�ͼ
	threshold(Lena, Lena, 100, 255, 0);
	image_show(Lena,1, "ԭͼ");

	//Mat X=Mat::zeros(Lena.size(),CV_8U);
	int M = 3;
	int N = 3;
	cv::Point p;//��image watch �鵽�ĵ�
	p.x = 93;
	p.y = 49;
	Mat X = Mat::zeros(Lena.size(), CV_8U);
	X.at<uchar>(p.y,p.x)=255;

	Point_gpu* point = set_Point_gpu(M, N);
	uchar* data = set_Point_data_pz(M, N);
	//char* file = "C:/Users/Administrator/Desktop/opencv/yuan.png";
	Mat mide = morphology_gpu_Mat(X, M*N, point, data, 1);
	Mat result = AND_NOT_two(mide, Lena, 0, 255);

	for (size_t i = 0; i < 20; i++)
	{ 	mide = morphology_gpu_Mat(result, M*N, point, data, 1);
		result = AND_NOT_two(mide, Lena, 0, 255);
	}

	result=OR_two(result, Lena);
	result.convertTo(result, CV_32F);
	image_show(result, 1, "���");
}

//ͳ��һ����ֵͼ�еĲ�Ϊ0�����ص����
int cout_image_thread(Mat& image,int max=255)
{   Scalar ss=sum(image);
	return (int)(ss[0] / max);
}

//��9.7
void connection_test() {
	Mat Lena = imread("C:/Users/Administrator/Desktop/opencv/chicken.png");
	cvtColor(Lena, Lena, COLOR_BGR2GRAY);//ת��Ϊ�Ҷ�ͼ
	threshold(Lena, Lena, 190, 255, 0);
	image_show(Lena, 1, "ԭͼ");

	//����ֻ��һ������û�н��и�ʴ����
	//Mat X=Mat::zeros(Lena.size(),CV_8U);
	int M = 3;
	int N = 3;
	cv::Point p;//��image watch �鵽�ĵ�
	p.x = 496;
	p.y = 89;
	Mat X = Mat::zeros(Lena.size(), CV_8U);
	X.at<uchar>(p.y, p.x) = 255;

	Point_gpu* point = set_Point_gpu(M, N);
	uchar* data = set_Point_data(M, N);//8��ͨ
	//char* file = "C:/Users/Administrator/Desktop/opencv/yuan.png";
	Mat mide = morphology_gpu_Mat(X, M*N, point, data, 1);
	Mat result = AND_two(mide, Lena, 0, 255);

	int mark = 0;
	for (size_t i = 0; i <1000; i++)
	{
		mide = morphology_gpu_Mat(result, M*N, point, data, 1);
		result = AND_two(mide, Lena, 0, 255);
		if (mark == cout_image_thread(result, 255))
			break;
		else
			mark = cout_image_thread(result, 255);
	}

	cout << "��ͨ���������ظ�����" <<mark<< endl;
    //result = OR_two(result, Lena);
	result.convertTo(result, CV_32F);
	image_show(result, 1, "��ͨ����ͼ");

}

//��������,�ȸ�ʴ��������,����ֵ�Ѿ��Ƕ�ֵ���Ժ��
Mat open_set(Mat& image,int M,int N,Point_gpu* point_N = NULL, uchar* data_N = NULL) {
	Mat Lena = image.clone();
	Lena.convertTo(Lena, CV_8U);

	Point_gpu* point;
	uchar* data;
	if (NULL != point_N && NULL!= data_N)
	{
		point = point_N;
		data = data_N;
	}
	else {
		point = set_Point_gpu(M, N);
		data = set_Point_data(M, N);
	}
	
	Mat mide = morphology_gpu_Mat(Lena, M*N, point, data, 0);
	mide = morphology_gpu_Mat(mide, M*N, point, data, 1);
	return mide.clone();
}

//�ռ�����,�����ͣ���ʴ,����ֵ�Ѿ��Ƕ�ֵ���Ժ��
Mat close_set(Mat& image, int M, int N, Point_gpu* point_N=NULL, uchar* data_N=NULL) {
	Mat Lena = image.clone();
	Lena.convertTo(Lena, CV_8U);

	Point_gpu* point;
	uchar* data;
	if (NULL != point_N && NULL != data_N)
	{  
		point = point_N;
		data = data_N;
	}
	else {
		point = set_Point_gpu(M, N);
		data = set_Point_data(M, N);
	}

	Mat mide = morphology_gpu_Mat(Lena, M*N, point, data, 1);
	mide = morphology_gpu_Mat(mide, M*N, point, data, 0);
	return mide.clone();
}

Mat bone_test() {
	Mat Lena = imread("C:/Users/Administrator/Desktop/opencv/fy.jpg");
	cvtColor(Lena, Lena, COLOR_BGR2GRAY);//ת��Ϊ�Ҷ�ͼ
	threshold(Lena, Lena, 100, 255, 0);
	Lena = NOT_two(Lena);
	//Mat Lena_close=Lena.clone();
	image_show(Lena, 1, "ԭͼ");

	int M = 3;
	int N = 3;
	Point_gpu* point;
	uchar* data;

	point = set_Point_gpu(M, N);
	data = set_Point_data(M, N);
	Mat vector[500];
	Mat AkB = Lena;
	int k = 500;

	for (size_t i = 0; i < k; i++)
	{
		vector[i] = Mat::zeros(Lena.size(),CV_8U);
	}
	
	for (size_t i = 0; i < k; i++)
	{
		AkB = morphology_gpu_Mat(AkB, M*N, point, data, 0);
		if (i == 10)
		{
			Mat Akb_N = AkB.clone();
			Akb_N.convertTo(Akb_N, CV_32F);
			image_show(Akb_N, 1, "AKB10");
		}
		vector[i]=AND_NOT_two(AkB, open_set(AkB, M, N, point, data));
		if (i == 10)
		{
			Mat Akb_m = vector[i].clone();
			Akb_m.convertTo(Akb_m, CV_32F);
			image_show(Akb_m, 1, "AKB*B10");
		}
	}
	
	Mat result= vector[k-1];
	for (size_t i = 0; i < k-1; i++)
	{
		result=OR_two(result, vector[i]);
	}

	result.convertTo(result, CV_32F);
	image_show(result, 1, "���");
	return result.clone();
}

//͹���������
void Prot_shell() {
	Mat Lena = imread("C:/Users/Administrator/Desktop/opencv/shell.png");
	cvtColor(Lena, Lena, COLOR_BGR2GRAY);//ת��Ϊ�Ҷ�ͼ
	threshold(Lena, Lena, 100, 255, 0);
	//Mat Lena_close=Lena.clone();
	image_show(Lena, 1, "ԭͼ");

	int M = 3;
	int N = 3;

	Point_gpu* point = set_Point_gpu(M, N);
	
	//B1
	uchar* B1 = set_Point_data_jz(M, N);
	B1[N * 0 + 0] = 255;
	B1[N * 1 + 0] = 255;
	B1[N * 2 + 0] = 255;

	//B2
	uchar* B2 = set_Point_data_jz(M, N);
	B2[N * 0 + 0] = 255;
	B2[N * 0 + 1] = 255;
	B2[N * 0 + 2] = 255;

	//B3
	uchar* B3 = set_Point_data_jz(M, N);
	B3[N * 0 + 2] = 255;
	B3[N * 1 + 2] = 255;
	B3[N * 2 + 2] = 255;

	//B4
	uchar* B4 = set_Point_data_jz(M, N);
	B4[N * 2 + 0] = 255;
	B4[N * 2 + 1] = 255;
	B4[N * 2 + 2] = 255;

	//B1 ����
	Mat X1=Lena.clone();
	Mat X1_N=Lena.clone();
	while (true){
		X1 = OR_two(Lena,morphology_gpu_Mat(X1, M*N, point, B1, 0));
		if (cout_image_thread(X1) == cout_image_thread(X1_N))
			break;
		else
			X1_N = X1.clone();
	}

	//B2 ����
	Mat X2 = Lena.clone();
	Mat X2_N = Lena.clone();
	while (true) {
		X2 = OR_two(Lena, morphology_gpu_Mat(X2, M*N, point, B2, 0));
		if (cout_image_thread(X2) == cout_image_thread(X2_N))
			break;
		else
			X2_N = X2.clone();
	}

	//B3 ����
	Mat X3 = Lena.clone();
	Mat X3_N = Lena.clone();
	while (true) {
		X3 = OR_two(Lena, morphology_gpu_Mat(X3, M*N, point, B3, 0));
		if (cout_image_thread(X3) == cout_image_thread(X3_N))
			break;
		else
			X3_N = X3.clone();
	}

	//B4 ����
	Mat X4 = Lena.clone();
	Mat X4_N = Lena.clone();
	while (true) {
		X4 = OR_two(Lena, morphology_gpu_Mat(X4, M*N, point, B4, 0));
		if (cout_image_thread(X4) == cout_image_thread(X4_N))
			break;
		else
			X4_N = X4.clone();
	}

	Mat result= OR_two(OR_two(OR_two(X1, X2), X3),X4);
	result.convertTo(result, CV_32F);
	image_show(result, 1, "���");
}

//�����ͱռ�
void  open_close_test()
{
	Mat Lena = imread("C:/Users/Administrator/Desktop/opencv/t.bmp");
	cvtColor(Lena, Lena, COLOR_BGR2GRAY);//ת��Ϊ�Ҷ�ͼ
	threshold(Lena, Lena, 10, 255, 0);
	//Mat Lena_close=Lena.clone();
	image_show(Lena, 1, "ԭͼ");

	//����ֻ��һ������û�н��и�ʴ����
	int M = 100;
	int N = 100;

	Mat mide = open_set(Lena,M,N);
	mide.convertTo(mide, CV_32F);
	image_show(mide, 1, "������");

	Mat mide_close=close_set(Lena,M,N);
	mide_close.convertTo(mide_close, CV_32F);
	image_show(mide_close, 1, "�ղ���");
}


//--------------------------------------�Ҷ���̬ͼ��ѧ------------------------------------
//�Ҷ�ͼ��ʴ-ƽ̹Բ
__device__ int change_center_gray(int x, int y,int imgHeight_des_d,int imgWidth_des_d,Point_gpu* point_gpu, int* data, int len) {
	int x_N;
	int y_N;
	int min=1000;

	for (int i = 0; i < len; i++)
	{
		x_N = (int)(point_gpu[i].x + x);
		y_N = (int)(point_gpu[i].y + y);
		if (data[i] == -255 && -1 < x_N && x_N < imgWidth_des_d && -1 < y_N && y_N < imgHeight_des_d)
		{   //����õ�data��i��=0,��ʾ���Ǹ�ʴ��������
			if ((int)tex2D(refTex_corrode, x_N, y_N) < min)
			   min = (int)tex2D(refTex_corrode, x_N, y_N);
		}
	}
	return  min;
}


//�Ҷ�ͼ��ʴ
__global__ void corrodeKerkel(int* pDstImgData, int imgHeight_des_d, int imgWidth_des_d, Point_gpu* point_gpu, int* data, int len)
{   //printf("threadIdx,x=%d",threadIdx.x);
	const int tidx = blockDim.x * blockIdx.x + threadIdx.x;
	const int tidy = blockDim.y * blockIdx.y + threadIdx.y;
	if (tidx < imgWidth_des_d && tidy < imgHeight_des_d)
	{
		int idx = tidy * imgWidth_des_d + tidx;
        pDstImgData[idx] = change_center_gray(tidx, tidy,imgHeight_des_d,imgWidth_des_d,point_gpu, data, len);
	}
}

//�Ҷ�ͼ������
__device__ int change_expand_gray(int x, int y, int imgHeight_des_d, int imgWidth_des_d, Point_gpu* point_gpu, int* data, int len) {
	int x_N;
	int y_N;
	int max = -265;

	for (int i = 0; i < len; i++)
	{
		x_N = (int)(point_gpu[i].x + x);
		y_N = (int)(point_gpu[i].y + y);
		if (data[i] == -255 && -1 < x_N && x_N < imgWidth_des_d && -1 < y_N && y_N < imgHeight_des_d)
		{   //����õ�data��i��=0,��ʾ���Ǹ�ʴ��������
			if ((int)tex2D(refTex_corrode, x_N, y_N) > max)
				max = (int)tex2D(refTex_corrode, x_N, y_N);
		}
	}
	return  max;
}

//�Ҷ�ͼ������
__global__ void expandKerkel(int* pDstImgData, int imgHeight_des_d, int imgWidth_des_d, Point_gpu* point_gpu, int* data, int len)
{   //printf("threadIdx,x=%d",threadIdx.x);
	const int tidx = blockDim.x * blockIdx.x + threadIdx.x;
	const int tidy = blockDim.y * blockIdx.y + threadIdx.y;
	if (tidx < imgWidth_des_d && tidy < imgHeight_des_d)
	{
		int idx = tidy * imgWidth_des_d + tidx;
        pDstImgData[idx] = change_expand_gray(tidx, tidy, imgHeight_des_d, imgWidth_des_d, point_gpu, data, len);
	}
}

//�Ҷ�ͼ��ʴ������
Mat morphology_gpu_gray(char * path, int len, Point_gpu*  point_offset_N, int* data, int mode) {
	se_tpye_gray * se_obj = (se_tpye_gray*)malloc(sizeof(se_tpye_gray));
	se_obj->init(len, point_offset_N, data);

	Mat Lena = imread(path);
	cvtColor(Lena, Lena, COLOR_BGR2GRAY);//ת��Ϊ�Ҷ�ͼ
	//threshold(Lena, Lena, 100, 255, 0);
	image_show(Lena, 1, "ԭͼ");

	int x_rato_less = 1.0;
	int y_rato_less = 1.0;

	int imgWidth_src = Lena.cols;//ԭͼ���
	int imgHeight_src = Lena.rows;//ԭͼ���
	int channels = Lena.channels();

	int imgWidth_des_less = floor(imgWidth_src * x_rato_less);//��Сͼ��
	int imgHeight_des_less = floor(imgHeight_src * y_rato_less);//��Сͼ��

	//����1��������
	hipError_t t;
	refTex_corrode.addressMode[0] = hipAddressModeBorder;
	refTex_corrode.addressMode[1] = hipAddressModeBorder;
	refTex_corrode.normalized = false;
	refTex_corrode.filterMode = hipFilterModePoint;
	//��cuArray������
	hipMallocArray(&cuArray_corrode, &cuDesc_corrode, imgWidth_src, imgHeight_src);
	t = hipBindTextureToArray(refTex_corrode, cuArray_corrode);
	//�������ݵ�hipArray
	t = hipMemcpyToArray(cuArray_corrode, 0, 0, Lena.data, imgWidth_src * imgHeight_src * sizeof(uchar), hipMemcpyHostToDevice);

	//��������Ժ���cpu��ͼ��
	Lena.convertTo(Lena, CV_32S);
	Mat dstImg1 = Mat::zeros(imgHeight_des_less, imgWidth_des_less, CV_32SC1);//��С

	//��������Ժ���cuda�ϵ�ͼ��
	int* pDstImgData1 = NULL;
	t = hipMalloc(&pDstImgData1, imgHeight_des_less * imgWidth_des_less * sizeof(int));
	t = hipMemcpy(pDstImgData1, Lena.data, imgWidth_des_less * imgHeight_des_less * sizeof(int)*channels, hipMemcpyHostToDevice);

	dim3 block(16, 16);
	dim3 grid((imgWidth_des_less + block.x - 1) / block.x, (imgHeight_des_less + block.y - 1) / block.y);

	if (mode == 0)
		corrodeKerkel << <grid, block >> > (pDstImgData1, imgHeight_des_less, imgWidth_des_less, se_obj->point_offset, se_obj->data, len);
	if (mode == 1)
		expandKerkel << <grid, block >> > (pDstImgData1, imgHeight_des_less, imgWidth_des_less, se_obj->point_offset, se_obj->data, len);
	hipDeviceSynchronize();

	//��GPU����������ݵ�CPU
	t = hipMemcpy(dstImg1.data, pDstImgData1, imgWidth_des_less * imgHeight_des_less * sizeof(int)*channels, hipMemcpyDeviceToHost);
	hipFree(cuArray_corrode);
	hipFree(pDstImgData1);
	stringstream ss;
	ss << len;
	string mark;
	ss >> mark;
	string ret;
	if (mode==1)
	   ret= string("����")+string("��ͼ") + mark;
	else
	   ret = string("��ʴ") + string("��ͼ") + mark;
	image_show(dstImg1, 1, ret.c_str());
	//namedWindow("cuda_point�����ֵ��", WINDOW_NORMAL);
	//imshow("��ʴ�Ժ��ͼ��", dstImg1);
	//imwrite("C:/Users/Administrator/Desktop/ͼƬ/Gray_Image0.jpg", dstImg1);
	return dstImg1.clone();
}

//�Ҷ�ͼ��ʴ������
Mat morphology_gpu_gray_Mat(Mat& Lena_N, int len, Point_gpu*  point_offset_N, int* data, int mode) {
	se_tpye_gray * se_obj = (se_tpye_gray*)malloc(sizeof(se_tpye_gray));
	se_obj->init(len, point_offset_N, data);

	Mat Lena = Lena_N.clone();
	//cvtColor(Lena, Lena, COLOR_BGR2GRAY);//ת��Ϊ�Ҷ�ͼ
	//threshold(Lena, Lena, 100, 255, 0);
	//image_show(Lena, 1, "ԭͼ");

	int x_rato_less = 1.0;
	int y_rato_less = 1.0;

	int imgWidth_src = Lena.cols;//ԭͼ���
	int imgHeight_src = Lena.rows;//ԭͼ���
	int channels = Lena.channels();

	int imgWidth_des_less = floor(imgWidth_src * x_rato_less);//��Сͼ��
	int imgHeight_des_less = floor(imgHeight_src * y_rato_less);//��Сͼ��

	//����1��������
	hipError_t t;
	refTex_corrode.addressMode[0] = hipAddressModeBorder;
	refTex_corrode.addressMode[1] = hipAddressModeBorder;
	refTex_corrode.normalized = false;
	refTex_corrode.filterMode = hipFilterModePoint;
	//��cuArray������
	hipMallocArray(&cuArray_corrode, &cuDesc_corrode, imgWidth_src, imgHeight_src);
	t = hipBindTextureToArray(refTex_corrode, cuArray_corrode);
	//�������ݵ�hipArray
	t = hipMemcpyToArray(cuArray_corrode, 0, 0, Lena.data, imgWidth_src * imgHeight_src * sizeof(uchar), hipMemcpyHostToDevice);

	//��������Ժ���cpu��ͼ��
	Lena.convertTo(Lena, CV_32S);
	Mat dstImg1 = Mat::zeros(imgHeight_des_less, imgWidth_des_less, CV_32SC1);//��С

	//��������Ժ���cuda�ϵ�ͼ��
	int* pDstImgData1 = NULL;
	t = hipMalloc(&pDstImgData1, imgHeight_des_less * imgWidth_des_less * sizeof(int));
	t = hipMemcpy(pDstImgData1, Lena.data, imgWidth_des_less * imgHeight_des_less * sizeof(int)*channels, hipMemcpyHostToDevice);

	dim3 block(16, 16);
	dim3 grid((imgWidth_des_less + block.x - 1) / block.x, (imgHeight_des_less + block.y - 1) / block.y);

	if (mode == 0)
		corrodeKerkel << <grid, block >> > (pDstImgData1, imgHeight_des_less, imgWidth_des_less, se_obj->point_offset, se_obj->data, len);
	if (mode == 1)
		expandKerkel << <grid, block >> > (pDstImgData1, imgHeight_des_less, imgWidth_des_less, se_obj->point_offset, se_obj->data, len);
	hipDeviceSynchronize();

	//��GPU����������ݵ�CPU
	t = hipMemcpy(dstImg1.data, pDstImgData1, imgWidth_des_less * imgHeight_des_less * sizeof(int)*channels, hipMemcpyDeviceToHost);
	hipFree(cuArray_corrode);
	hipFree(pDstImgData1);
	//stringstream ss;
	//ss << len;
	//string mark;
	//ss >> mark;
	//string ret;
	//if (mode == 1)
	//	ret = string("����") + string("��ͼ") + mark;
	//else
	//	ret = string("��ʴ") + string("��ͼ") + mark;
	//image_show(dstImg1, 1, ret.c_str());
	//namedWindow("cuda_point�����ֵ��", WINDOW_NORMAL);
	//imshow("��ʴ�Ժ��ͼ��", dstImg1);
	//imwrite("C:/Users/Administrator/Desktop/ͼƬ/Gray_Image0.jpg", dstImg1);
	return dstImg1.clone();
}

Mat open_set_gray(Mat& image, int M, int N, Point_gpu* point_N = NULL, int* data_N = NULL) {
	Mat Lena = image.clone();
	Lena.convertTo(Lena, CV_8U);

	Point_gpu* point;
	int* data;
	if (NULL != point_N && NULL != data_N)
	{
		point = point_N;
		data = data_N;
	}
	else {
		point = set_Point_gpu(M, N);
		data = set_Point_data_circle(M);
	}

	Mat mide = morphology_gpu_gray_Mat(Lena, M*N, point, data, 0);
	mide.convertTo(mide, CV_8U);
	mide = morphology_gpu_gray_Mat(mide, M*N, point, data, 1);
	return mide.clone();
}

//�ռ�����,�����ͣ���ʴ,����ֵ�Ѿ��Ƕ�ֵ���Ժ��
Mat close_set_gray(Mat& image, int M, int N, Point_gpu* point_N = NULL, int* data_N = NULL) {
	Mat Lena = image.clone();
	Lena.convertTo(Lena, CV_8U);

	Point_gpu* point;
	int* data;
	if (NULL != point_N && NULL != data_N)
	{
		point = point_N;
		data = data_N;
	}
	else {

		point = set_Point_gpu(M, N);
		data = set_Point_data_circle(M);
	}

	Mat mide = morphology_gpu_gray_Mat(Lena, M*N, point, data, 1);
	mide.convertTo(mide, CV_8U);
	mide = morphology_gpu_gray_Mat(mide, M*N, point, data, 0);
	return mide.clone();
}

///�Ҷ���̬ͼ��ѧ ����
void gray_test() {

	int M = 3;
	int N = 3;
	Point_gpu* point = set_Point_gpu(M, N);
	int* data = set_Point_data_circle(M);

	//morphology_gpu_gray("C:/Users/Administrator/Desktop/opencv/dl.png", M*N, point, data, 0);
	//morphology_gpu_gray("C:/Users/Administrator/Desktop/opencv/dl.png", M*N, point, data, 1);
	Mat Lena = imread("C:/Users/Administrator/Desktop/opencv/dl.png");
	cvtColor(Lena, Lena, COLOR_BGR2GRAY);//ת��Ϊ�Ҷ�ͼ
	image_show(Lena,1,"ԭͼ");
	Mat close=close_set_gray(Lena, M, N, NULL, NULL);
	image_show(close, 1,"close");
	Mat open=open_set_gray(Lena, 10, 10, NULL, NULL);
	image_show(open, 1,"open");
}

void chapter9() {
	//test();//����morphology_gpu�Ƿ���ȷ
	//morphology_test(5, 5, 0);//����9.1
	//morphology_test(3, 3, 1);//����9.1
	//man_test();//����9.5
	//remove_test();//����9.6
	//connection_test();
	//open_close_test();
	//bone_test();
	//Prot_shell();
	gray_test();
}


//---------------------------------------------��ʮ��----------------------------------
texture <uchar, hipTextureType2D, hipReadModeElementType> refTex_space_filter;//���ڼ���˫���Բ�ֵ

hipArray* cuArray_space_filter;//����CUDA����

//ͨ����
hipChannelFormatDesc cuDesc_space_filter = hipCreateChannelDesc<uchar>();//ͨ����

////�ռ��˲�
__device__ int spacefilter(int x, int y, Point_gpu* point_gpu, int* data, int len) {
	int x_N;
	int y_N;
	int result = 0;
	for (int i = 0; i < len; i++)
	{
		x_N = (int)(point_gpu[i].x + x);
		y_N = (int)(point_gpu[i].y + y);
		result = result + (int)((float)tex2D(refTex_space_filter, x_N, y_N)*data[i]);
		/*	if (x == 0 && y == 0)
				printf("x:%d,y:%d,|%d,%d,%d,%d \n",x_N,y_N, i, (int)((int)tex2D(refTex_space_filter, x_N, y_N)*data[i]),(int)tex2D(refTex_space_filter, x_N, y_N),data[i]);*/
	}
	return  result;
}

////�ռ��˲�
__global__ void space_filter_Kerkel(int* pDstImgData, int imgHeight_des_d, int imgWidth_des_d, Point_gpu* point_gpu, int* data, int len)
{   //printf("threadIdx,x=%d",threadIdx.x);
	const int tidx = blockDim.x * blockIdx.x + threadIdx.x;
	const int tidy = blockDim.y * blockIdx.y + threadIdx.y;
	if (tidx < imgWidth_des_d && tidy < imgHeight_des_d)
	{
		int idx = tidy * imgWidth_des_d + tidx;
		pDstImgData[idx] = spacefilter(tidx, tidy, point_gpu, data, len);
	}
}

Mat space_filter_cpu(char * path, int len, Point_gpu*  point_offset_N, int* data, float size)
{
	Mat Lena = imread(path);
	cvtColor(Lena, Lena, COLOR_BGR2GRAY);//ת��Ϊ�Ҷ�ͼ
	image_show(Lena, size, "ԭͼ");

	int x_rato_less = 1.0;
	int y_rato_less = 1.0;

	int imgWidth_src = Lena.cols;//ԭͼ���
	int imgHeight_src = Lena.rows;//ԭͼ���
	int channels = Lena.channels();

	int imgWidth_des_less = floor(imgWidth_src * x_rato_less);//��Сͼ��
	int imgHeight_des_less = floor(imgHeight_src * y_rato_less);//��Сͼ��

	//����1��������
	hipError_t t;
	refTex_space_filter.addressMode[0] = hipAddressModeClamp;
	refTex_space_filter.addressMode[1] = hipAddressModeClamp;
	refTex_space_filter.normalized = false;
	refTex_space_filter.filterMode = hipFilterModePoint;
	//��cuArray������
	hipMallocArray(&cuArray_space_filter, &cuDesc_space_filter, imgWidth_src, imgHeight_src);
	t = hipBindTextureToArray(refTex_space_filter, cuArray_space_filter);
	//�������ݵ�hipArray
	t = hipMemcpyToArray(cuArray_space_filter, 0, 0, Lena.data, imgWidth_src * imgHeight_src * sizeof(uchar), hipMemcpyHostToDevice);

	//��������Ժ���cpu��ͼ��
	Lena.convertTo(Lena, CV_32S);
	//cout<<Lena<<endl;
	Mat dstImg1 = Mat::zeros(imgHeight_des_less, imgWidth_des_less, CV_32SC1);//��С

	//��������Ժ���cuda�ϵ�ͼ��
	int* pDstImgData1 = NULL;
	t = hipMalloc(&pDstImgData1, imgHeight_des_less * imgWidth_des_less * sizeof(int));
	t = hipMemcpy(pDstImgData1, Lena.data, imgWidth_des_less * imgHeight_des_less * sizeof(int)*channels, hipMemcpyHostToDevice);

	dim3 block(16, 16);
	dim3 grid((imgWidth_des_less + block.x - 1) / block.x, (imgHeight_des_less + block.y - 1) / block.y);

	space_filter_Kerkel << <grid, block >> > (pDstImgData1, imgHeight_des_less, imgWidth_des_less, point_offset_N, data, len);
	hipDeviceSynchronize();

	//��GPU����������ݵ�CPU
	t = hipMemcpy(dstImg1.data, pDstImgData1, imgWidth_des_less * imgHeight_des_less * sizeof(int)*channels, hipMemcpyDeviceToHost);
	hipFree(cuArray_space_filter);
	hipFree(pDstImgData1);
	//namedWindow("cuda_point�����ֵ��", WINDOW_NORMAL);
	//imshow("��ʴ�Ժ��ͼ��", dstImg1);
	//imwrite("C:/Users/Administrator/Desktop/ͼƬ/Gray_Image0.jpg", dstImg1);
	//Lena = Lena - dstImg1;
	//Lena.convertTo(Lena, CV_8U);
	//imshow("���򱱼�_������˹�任���ͼ", Lena);
	//result.convertTo(result, CV_32F);
	//image_show(Lena, 1, "�仯���ͼ");
	return dstImg1.clone();
}

//��������
void single_point()
{
	int M = 3;
	int N = 3;
	filter_screem* filter = (filter_screem*)malloc(sizeof(filter_screem));
	filter->init(M, N);
	filter->data[4] = -8;
	Mat result = space_filter_cpu("C:/Users/Administrator/Desktop/opencv/point.png", filter->len, filter->postion, filter->data, 2);
	result = abs(result);
	result.convertTo(result, CV_8U);
	threshold(result, result, 140, 255, 0);
	image_show(result, 2, "������ͼ");
}

//ֱ�߼��
void line_test() {
	//ˮƽ
	int M = 3;
	int N = 3;
	filter_screem* filter_sp = (filter_screem*)malloc(sizeof(filter_screem));
	filter_sp->init(M, N);
	int src[9] = { -1,-1,-1,2,2,2,-1,-1,-1 };
	hipMemcpy(filter_sp->data, src, sizeof(int)*N*M, hipMemcpyDefault);
	Mat result = space_filter_cpu("C:/Users/Administrator/Desktop/opencv/m486.png", filter_sp->len, filter_sp->postion, filter_sp->data, 0.4);
	result.convertTo(result, CV_8U);
	threshold(result, result, 20, 255, 0);
	image_show(result, 0.4, "ˮƽ����ȡ");

	//��ֱ
	filter_sp = (filter_screem*)malloc(sizeof(filter_screem));
	filter_sp->init(M, N);
	int src_cz[9] = { -1,2,-1,-1,2,-1,-1,2,-1 };
	hipMemcpy(filter_sp->data, src_cz, sizeof(int)*N*M, hipMemcpyDefault);
	result = space_filter_cpu("C:/Users/Administrator/Desktop/opencv/m486.png", filter_sp->len, filter_sp->postion, filter_sp->data, 0.4);
	result.convertTo(result, CV_8U);
	threshold(result, result, 20, 255, 0);
	image_show(result, 0.4, "��ֱ����ȡ");

	//+45��
	filter_sp = (filter_screem*)malloc(sizeof(filter_screem));
	filter_sp->init(M, N);
	int src_z45[9] = { 2,-1,-1,-1,2,-1,-1,-1,2 };
	hipMemcpy(filter_sp->data, src_z45, sizeof(int)*N*M, hipMemcpyDefault);
	result = space_filter_cpu("C:/Users/Administrator/Desktop/opencv/m486.png", filter_sp->len, filter_sp->postion, filter_sp->data, 0.4);
	result.convertTo(result, CV_8U);
	threshold(result, result, 10, 255, 0);
	image_show(result, 0.4, "45��+����ȡ");

	//-45��
	filter_sp = (filter_screem*)malloc(sizeof(filter_screem));
	filter_sp->init(M, N);
	int src_f45[9] = { -1,-1,2,-1,2,-1,2,-1,-1 };
	hipMemcpy(filter_sp->data, src_f45, sizeof(int)*N*M, hipMemcpyDefault);
	result = space_filter_cpu("C:/Users/Administrator/Desktop/opencv/m486.png", filter_sp->len, filter_sp->postion, filter_sp->data, 0.4);
	result.convertTo(result, CV_8U);
	threshold(result, result, 10, 255, 0);
	image_show(result, 0.4, "45��-����ȡ");
}

//10.2.5������Ե���
//��10.6
enum spacefilter_mode {
	prewitt_x = 0,
	prewitt_y = 1,

    sobel_x = 2,
	sobel_y =3,

	sobel_45z = 4,
	sobel_45f = 5,
};

filter_screem* set_filter(spacefilter_mode mode){
  filter_screem* filter = (filter_screem*)malloc(sizeof(filter_screem));
  int M = 3;
  int N = 3;
  filter->init(M, N);

  if (mode == 0)
  {
	  int src[9] = { -1,0,1,-1,0,1,-1,0,1 };
	  hipMemcpy(filter->data, src, sizeof(int)*N*M, hipMemcpyDefault);
  }

  if (mode == 1)
  {
	  int src[9] = { -1,-1,-1,0,0,0,1,1,1 };
	  hipMemcpy(filter->data, src, sizeof(int)*N*M, hipMemcpyDefault);
  }

  if (mode == 2)
  {
	  int src[9] = { -1,0,1,-2,0,2,-1,0,1 };
	  hipMemcpy(filter->data, src, sizeof(int)*N*M, hipMemcpyDefault);
  }


  if (mode == 3)
  {
	  int src[9] = { -1,-2,-1,0,0,0,1,2,1};
	  hipMemcpy(filter->data, src, sizeof(int)*N*M, hipMemcpyDefault);
  }

  if (mode == 4)
  {
	  int src[9] = { 0,1,2,-1,0,1,-2,-1,0 };
	  hipMemcpy(filter->data, src, sizeof(int)*N*M, hipMemcpyDefault);
  }

  if (mode == 5)
  {
	  int src[9] = { -2,-1,0,-1,0,1,0,1,2 };
	  hipMemcpy(filter->data, src, sizeof(int)*N*M, hipMemcpyDefault);
  }

  return filter;
}

void two_fd_jd_test()
{
	Mat Lena = imread("C:/Users/Administrator/Desktop/opencv/house.png");
	cvtColor(Lena, Lena, COLOR_BGR2GRAY);//ת��Ϊ�Ҷ�ͼ
	image_show(Lena, 1, "ԭͼ");
	
	filter_screem* filter_x = set_filter(sobel_x);
	Mat result_x = space_filter_cpu("C:/Users/Administrator/Desktop/opencv/house.png", filter_x->len, filter_x->postion, filter_x->data, 1);
	//cout << result_x << endl;
	
	filter_screem* filter_y = set_filter(sobel_y);
	Mat result_y = space_filter_cpu("C:/Users/Administrator/Desktop/opencv/house.png", filter_y->len, filter_y->postion, filter_y->data, 1);
	
	filter_screem* filter_45z = set_filter(sobel_45z);
	Mat result_45z = space_filter_cpu("C:/Users/Administrator/Desktop/opencv/house.png", filter_45z->len, filter_45z->postion, filter_45z->data, 1);

	filter_screem* filter_45f = set_filter(sobel_45f);
	Mat result_45f = space_filter_cpu("C:/Users/Administrator/Desktop/opencv/house.png", filter_45f->len, filter_45f->postion, filter_45f->data, 1);


	Mat xy = result_x + result_y;
	xy.convertTo(xy, CV_8U);
	image_show(xy, 1, "sobel_y+soble_y");

	Mat xy_abs = abs(result_x) + abs(result_y);
	xy_abs.convertTo(xy_abs, CV_8U);
	//imshow("abs:sobel_y+soble_y",xy_abs);
	image_show(xy_abs, 1, "abs:sobel_y+soble_y");
	
	result_x=abs(result_x);
	result_x.convertTo(result_x, CV_8U);
	//imshow("sobel_x", result_x);
	image_show(result_x, 1, "sobel_x");

	result_y = abs(result_y);
	result_y.convertTo(result_y, CV_8U);
	//imshow("sobel_y", result_y);
	image_show(result_y, 1, "sobel_y");

	result_45z = abs(result_45z);
	result_45z.convertTo(result_45z, CV_8U);
	image_show(result_45z, 1, "sobel_45z");

	result_45f = abs(result_45f);
	result_45f.convertTo(result_45f, CV_8U);
	image_show(result_45f, 1, "sobel_45f");
}


void chapter10_test()
{
	//single_point();
	//line_test();
	two_fd_jd_test();
};